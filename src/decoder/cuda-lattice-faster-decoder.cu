#include "hip/hip_runtime.h"
// decoder/simple-decoder.cc

// Copyright      2018  Zhehuai Chen

// See ../../COPYING for clarification regarding multiple authors
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//  http://www.apache.org/licenses/LICENSE-2.0
//
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.

#include "decoder/cuda-decoder.h"
#include "fstext/remove-eps-local.h"
#include <algorithm>
#include <nvToolsExt.h>
#include <hip/hip_runtime_api.h>
#include <float.h>
#include <math.h>

#include <hipcub/hipcub.hpp>

#define MEMADVISE

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
        hipError_t e=hipGetLastError();                                 \
        if(e!=hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    }

#define DIV_ROUND_UP(a,b) ((a+b-1)/b)
#define COMPUTE_DEGREES_DIMX 64
#define EXPAND_ARCS_DIMX 64
#define NONEM_LT_DIMX 1024
// Below that value, we launch the persistent kernel for NonEmitting
#define NONEM_LT_MAX_NARCS (4*NONEM_LT_DIMX) //4096
namespace kaldi {

// for speedup purpose, make them inline (5% 0.165->0.158)
inline HOST DEVICE uint64_t pack (float cost, int ptr) {
  // assert (!isnan(cost));
  // assert (ptr >= 0 && ptr < 1L<<32);
  uint32_t i_cost = *(uint32_t *)&cost;
  if (i_cost & 0x80000000)
    i_cost = i_cost ^ 0xFFFFFFFF;
  else
    i_cost = i_cost ^ 0x80000000;
  return (uint64_t)i_cost << 32 | ptr;
}

// Unpacks a probability.
inline HOST DEVICE float unpack_cost (uint64_t packed) {
  uint32_t i_cost = packed >> 32;
  if (i_cost & 0x80000000)
    i_cost = i_cost ^ 0x80000000;
  else
    i_cost = i_cost ^ 0xFFFFFFFF;
  return *(float *)&i_cost;
}

// Unpacks a back-pointer.
inline HOST DEVICE int unpack_ptr (uint64_t packed) {
  // assert (!(packed & 0x80000000));
  return packed & 0x7FFFFFFF;
}

// Used to trigger the fire&forget version of atomicMin (only av for int/long)
HOST DEVICE uint floatToOrderedUInt(float floatVal) {
    uint i_cost = *(uint*)( &floatVal );
  if (i_cost & 0x80000000)
    i_cost = i_cost ^ 0xFFFFFFFF;
  else
    i_cost = i_cost ^ 0x80000000;
  return i_cost;

}



__host__ __device__ float orderedUIntToFloat(uint i_cost) {
  if (i_cost & 0x80000000)
    i_cost = i_cost ^ 0x80000000;
  else
    i_cost = i_cost ^ 0xFFFFFFFF;
  return *(BaseFloat *) & i_cost;
} 


// Assumptions: 1-d grid and blocks. No threads "early-exit" the grid.
// No stream priorities
static DEVICE inline void _grid_sync(volatile int *fast_epoch) {
  __syncthreads();
  if (threadIdx.x == 0) {
    // gridDim.x-1 blocks are adding 1
    // and one block is adding 0x80000000 - (gridDim.x-1)
    // so the whole sum is 0x80000000
    int nb = 1;
    if (blockIdx.x == 0) {
      nb = 0x80000000 - (gridDim.x - 1);
    }
    int old_epoch = *fast_epoch;
    __threadfence();
    atomicAdd((int*)fast_epoch, nb);
    // wait for the sign bit to commute
    int cnt = 0;
    while (((*fast_epoch) ^ old_epoch) >= 0) ;
  }
  __syncthreads();
}

DEVICE inline void grid_sync(int *barrier) {
  _grid_sync((volatile int*)barrier);
}



  /***************************************CudaFst Implementation*****************************************/
  HOST DEVICE inline float CudaFst::Final(StateId state) const {
    #ifdef __CUDA_ARCH__
    return final_d[state];
    #else
    return final_h[state];
    #endif

  }
  void CudaFst::initialize(const fst::Fst<StdArc> &fst) {
    nvtxRangePushA("CudaFst constructor");
    bytes_cudaMalloc=0;
    //count states since Fst doesn't provide this functionality
    numStates=0;
    for( fst::StateIterator<fst::Fst<StdArc> > iter(fst); !iter.Done(); iter.Next()) {
      numStates++;
    }
    start=fst.Start();
    hipHostMalloc(&final_h,sizeof(float)*numStates);
    hipMalloc(&final_d,sizeof(float)*numStates);

    //allocate and initialize offset arrays
    e_offsets_h=(unsigned int *)malloc(sizeof(unsigned int)*(numStates+1));
    ne_offsets_h=(unsigned int *)malloc(sizeof(unsigned int)*(numStates+1));

    hipMalloc((void**)&e_offsets_d,sizeof(unsigned int)*(numStates+1)); bytes_cudaMalloc+=sizeof(unsigned int)*(numStates+1);
    hipMalloc((void**)&ne_offsets_d,sizeof(unsigned int)*(numStates+1)); bytes_cudaMalloc+=sizeof(unsigned int)*(numStates+1);

    memset(e_offsets_h,0,sizeof(unsigned int)*(numStates+1));
    memset(ne_offsets_h,0,sizeof(unsigned int)*(numStates+1));

    //iterate through states and arcs and count number of arcs per state
    e_count=0;
    ne_count=0;
    max_ilabel=0;

    for(int i=0;i<numStates;i++) {
      final_h[i]=fst.Final(i).Value();
      //count emmiting and non_emitting arcs
      for (fst::ArcIterator<fst::Fst<StdArc> > aiter(fst, i); !aiter.Done(); aiter.Next()) {
        StdArc arc = aiter.Value();
        int32 ilabel = arc.ilabel;
        int32 olabel = arc.olabel;

        if(ilabel>max_ilabel) {
          max_ilabel=ilabel;
        }

        if(ilabel!=0) { //emitting
          e_count++;
        } else { //non-emitting
          ne_count++;
        }
      }
      ne_offsets_h[i+1]=ne_count;
      e_offsets_h[i+1]=e_count;
    }

    //offset ne_offsets by the number of emitting arcs
    for(int i=0;i<numStates+1;i++) {
      e_offsets_h[i]+=1;          //add dummy arc at the beginingg.
      ne_offsets_h[i]+=e_count+1;   //add dummy arc and put e_arcs before
    }

    arc_count=e_count+ne_count+1;

    hipMemcpy(final_d,final_h,sizeof(float)*numStates,hipMemcpyHostToDevice);
    
    hipMemcpy(e_offsets_d,e_offsets_h,sizeof(unsigned int)*(numStates+1),hipMemcpyHostToDevice);
    hipMemcpy(ne_offsets_d,ne_offsets_h,sizeof(unsigned int)*(numStates+1),hipMemcpyHostToDevice);


    //Allocate non-zero arrays
    hipHostMalloc(&arc_weights_h,arc_count*sizeof(BaseFloat));
    hipHostMalloc(&arc_nextstates_h,arc_count*sizeof(StateId));
    hipHostMalloc(&arc_ilabels_h,arc_count*sizeof(int32));
    hipHostMalloc(&arc_olabels_h,arc_count*sizeof(int32));

    hipMalloc((void**)&arc_weights_d,arc_count*sizeof(BaseFloat));
    hipMalloc((void**)&arc_nextstates_d,arc_count*sizeof(StateId));
    hipMalloc((void**)&arc_ilabels_d,arc_count*sizeof(int32)); 

        //now populate arc data
    int e_idx=1;          //save room for dummy arc (so start at 1)
    int ne_idx=e_count+1; //starts where e_offsets ends

    //create dummy arc
    arc_weights_h[0]=StdWeight::One().Value();
    arc_nextstates_h[0]=fst.Start();
    arc_ilabels_h[0]=0;
    arc_olabels_h[0]=0;

    for(int i=0;i<numStates;i++) {
      //count emiting and non_emitting arcs

      for (fst::ArcIterator<fst::Fst<StdArc> > aiter(fst, i); !aiter.Done(); aiter.Next()) {
        StdArc arc = aiter.Value();
        int idx;
        if(arc.ilabel!=0) { //emitting
          idx=e_idx++;
        } else {
          idx=ne_idx++;
        }
        arc_weights_h[idx]=arc.weight.Value();
        arc_nextstates_h[idx]=arc.nextstate;
        arc_ilabels_h[idx]=arc.ilabel;
        arc_olabels_h[idx]=arc.olabel;
      }
    }

    hipMemcpy(arc_weights_d,arc_weights_h,arc_count*sizeof(BaseFloat),hipMemcpyHostToDevice);
    hipMemcpy(arc_nextstates_d,arc_nextstates_h,arc_count*sizeof(StateId),hipMemcpyHostToDevice);
    hipMemcpy(arc_ilabels_d,arc_ilabels_h, arc_count*sizeof(int32),hipMemcpyHostToDevice);
    
    hipDeviceSynchronize();
    cudaCheckError();

    nvtxRangePop();
  }

  void CudaFst::finalize() {
    nvtxRangePushA("CudaFst destructor");
    hipHostFree(final_h);
    hipFree(final_d);
    free(e_offsets_h);
    free(ne_offsets_h);

    hipFree(e_offsets_d);
    hipFree(ne_offsets_d);

    hipHostFree(arc_weights_h);
    hipHostFree(arc_nextstates_h);
    hipHostFree(arc_ilabels_h);
    hipHostFree(arc_olabels_h);

    hipFree(arc_weights_d);
    hipFree(arc_nextstates_d);
    hipFree(arc_ilabels_d);
    nvtxRangePop();
  }

  /***************************************End CudaFst****************************************************/


// LatticeProcessor Implementation
// Initialize in InitDecoding()
void LatticeProcessor::Initialize() {
  hipMemset(arcs_apr_fr_size_d, 0, sizeof(int32) * (prune_interval + 2));
  hipMemset(arcs_apr_used_d, 0, sizeof(int32));
  hipMemset(arcs_bpr_used_d, 0, sizeof(int32));
  hipMemset(toks_bpr_fr_sidx_d, 0, sizeof(int32) * (prune_interval + 2));
  hipMemset(arcs_bpr_fr_sidx_d, 0, sizeof(int32) * (prune_interval + 2));
  hipMemset(toks_num_used, 0, sizeof(int32));
}

// the return value including the hipMallocManaged size
int32 LatticeProcessor::Allocate(int32 max_tokens_per_frame,
                              int32 max_lat_arc_per_frame, int32 prune_interval,
                              int32 max_toks, int32 max_arcs,
                              const CudaFst& fst) {
  int32 sz;
  int32 bytes_cuda_malloc = 0;

  // before pruning
  // to reduce memory usage, we use hipMallocManaged, which doesn't
  // allocate in GPU at once
  sz = sizeof(Token) * max_toks;
  cuda_malloc_managed_preferred_device((void**)&toks_bpr_d, sz);
  bytes_cuda_malloc += sz;
  // if we directly use managed memory from toks_bpr_d, the RTF is 10% larger
  hipHostMalloc((void**)&toks_bpr_h, sz);
  toks_buf_before_pr_size = sz / sizeof(Token);

  // to reduce memory usage, we use hipMallocManaged, which doesn't
  // allocate in GPU at once
  sz = sizeof(LatLinkCompact) * max_arcs;
  cuda_malloc_managed_preferred_device((void**)&arcs_bpr_d, sz);
  bytes_cuda_malloc += sz;

  arcs_buf_before_pr_size = max_arcs;
  sz = sizeof(int32) * (prune_interval + 2);
  hipMalloc((void**)&toks_bpr_fr_sidx_d, sz); bytes_cuda_malloc += sz;
  hipHostMalloc((void**)&toks_bpr_fr_sidx_h, sz);
  sz = sizeof(int32);
  hipMalloc((void**)&toks_num_used, sz); bytes_cuda_malloc += sz;
  sz = sizeof(int32) * (prune_interval + 2);
  hipMalloc((void**)&arcs_bpr_fr_sidx_d, sz); bytes_cuda_malloc += sz;

  // after pruning
  sz = sizeof(int32) * (prune_interval + 2);
  hipMalloc((void**)&arcs_apr_fr_size_d, sz); bytes_cuda_malloc += sz;
  hipHostMalloc((void**)&arcs_apr_fr_size_h, sz);
  sz = ESTIMATED_PRUNE_RATIO * sizeof(LatLink) * max_arcs;
  // to reduce memory usage, we use hipMallocManaged, which doesn't
  // allocate in GPU at once
  cuda_malloc_managed_preferred_device((void**)&arcs_apr_d, sz);
  bytes_cuda_malloc += sz;
  hipHostMalloc((void**)&arcs_apr_h, sz);
  sz = sizeof(int32);
  hipMalloc((void**)&arcs_apr_used_d, sz); bytes_cuda_malloc += sz;
  hipMalloc((void**)&arcs_bpr_used_d, sz); bytes_cuda_malloc += sz;
  hipHostMalloc((void**)&arcs_apr_used_h, sz);

  // GPU global memory temp variables
  sz = sizeof(int32);
  hipMalloc((void**)&barrier_, sz); bytes_cuda_malloc += sz;
  sz = sizeof(int32) * 3;
  hipMalloc((void**)&modified_d, sz); bytes_cuda_malloc += sz;
  sz = sizeof(int32) * (2);
  hipMalloc((void**)&count_vec_acc_d, sz); bytes_cuda_malloc += sz;
  this->prune_interval = prune_interval;

  arc_ilabels = fst.arc_ilabels_d;
  arc_olabels = fst.arc_olabels_d;
  arc_weights = fst.arc_weights_d;
  return bytes_cuda_malloc;
}
void LatticeProcessor::Free() {
  // before pruning
  hipFree(arcs_bpr_used_d);
  hipHostFree(arcs_apr_used_h);
  //hipFree(toks_bpr_d);
  hipHostFree(toks_bpr_h);
  hipFree(arcs_bpr_d);
  hipFree(toks_bpr_fr_sidx_d);
  hipHostFree(toks_bpr_fr_sidx_h);
  hipFree(arcs_bpr_fr_sidx_d);
  hipFree(toks_num_used);

  // after pruning
  hipFree(arcs_apr_fr_size_d);
  hipHostFree(arcs_apr_fr_size_h);
  hipFree(arcs_apr_d);
  hipFree(arcs_apr_used_d);

  // GPU global memory temp variables
  hipFree(count_vec_acc_d);
  hipFree(barrier_);
  hipFree(modified_d);
  hipHostFree(arcs_apr_h);
}

DEVICE Token* LatticeProcessor::GetTokenByExactIdx(uint32 offset) {
  int32 idx = offset;
#ifdef __DEBUG__
  assert(idx >= 0 && idx < toks_buf_before_pr_size);
#else
  if (idx >= toks_buf_before_pr_size) idx = toks_buf_before_pr_size - 1;
#endif
  return toks_bpr_d + idx;
}

DEVICE int32 LatticeProcessor::GetTokenAllocIdx(uint32 offset) {
  int32 idx = *toks_num_used + offset;
#ifdef __DEBUG__
  assert(idx >= 0 && idx < toks_buf_before_pr_size);
#else
  if (idx >= toks_buf_before_pr_size) idx = toks_buf_before_pr_size - 1;
#endif
  return idx;
}

DEVICE int32 LatticeProcessor::GetTokenIdxFromAddr(Token* tok) {
  int32 ret = tok - toks_bpr_d;
  assert(ret < toks_buf_before_pr_size && ret >= 0);
  return ret;
}

// entry of lattice pruning until this frame
DEVICE void LatticeProcessor::PruneActiveTokens(int32 frame,
    BaseFloat lattice_beam, int32 verbose) {
  int32 rank0 = threadIdx.x == 0 && blockIdx.x == 0 ? 1 : 0;
  if (frame == 0) return;
  if (rank0) *arcs_apr_used_d = 0; // clear buffer index
  grid_sync(barrier_);
  for (int32 f = frame; f > 0; f--) { // prune each frame in serial
    PruneLatticeForFrame(f, 1, lattice_beam, verbose);
  }
  // by ESTIMATED_PRUNE_RATIO to reduce memory allocation and D2H data transfer
  assert(*arcs_apr_used_d < arcs_buf_before_pr_size * ESTIMATED_PRUNE_RATIO);
  if (verbose > 2 && rank0)
    CUDA_PRINTF("PRt: %i %i\n", arcs_bpr_fr_sidx_d[frame + 1],
                *arcs_apr_used_d);
}

// collect after each token passing, we store Token data in the sequence of
// TokenState vector, using continuous memory
DEVICE void LatticeProcessor::CollectToksPerFrame(int *cur_size, int32 frame) {
  int32 tid = threadIdx.x + blockIdx.x * blockDim.x;
  int32 size = *cur_size - *toks_num_used;
  if (tid == 0) {
    // Set start index in the buffer of the next frame
    SetNextSidx(toks_bpr_fr_sidx_d, size, frame);
    *toks_num_used = *cur_size;
    assert(*toks_bpr_fr_sidx_d < toks_buf_before_pr_size);
  }
}

// collect after each token passing, mainly to update arcs_bpr_fr_sidx_d here
DEVICE void LatticeProcessor::CollectArcsPerFrame(int *cur_size, int32 frame) {
  int32 idx = threadIdx.x + blockIdx.x * blockDim.x;
  int32 rank0 = blockIdx.x == 0 && threadIdx.x == 0 ? 1 : 0;
  int32 batch = blockDim.x * gridDim.x;
  int32 size = *cur_size - *arcs_bpr_used_d; // size of current frame
  if (rank0) {
    SetNextSidx(arcs_bpr_fr_sidx_d, size, frame);
    *arcs_bpr_used_d = *cur_size;
    // we didn't clear cur_arc_array.count_d until the end of decoding
  }
  /*
  // we share the memory between vector&pruner, so dont need to copy between them
  for(; idx < size; idx += batch) {
    LatLink* to_arc=GetActiveArc(frame,(idx));
    fast_store32(to_arc, cur_arc_array.mem_d+idx);
    // for debug purpose
    GetActiveToken((cur_arc_array.mem_d+idx)->p1,true,frame);
    GetActiveToken(to_arc->p1,true,frame);
  }
  */
}

// AddArc function implemented
// by an atomic operation, where the memory is pre-allocated
DEVICE int32 LatticeProcessor::AddArc(LatLink* arc) {
  int32 i = atomicAdd(arcs_apr_used_d, 1);
  assert(i < arcs_buf_before_pr_size * ESTIMATED_PRUNE_RATIO);
  fast_store32(arcs_apr_d + i, arc);
}
DEVICE int32 LatticeProcessor::AddArc(LatLinkCompact* arc, int32 frame) {
  int32 i = atomicAdd(arcs_apr_used_d, 1);
  assert(i < arcs_buf_before_pr_size * ESTIMATED_PRUNE_RATIO);
  int32 frame_tok = arc->IsEmitArc() ? frame - 1 : frame;
  int32 j = arc->arc_id;
  LatLink apr_arc(arc->GetPrevTokId(), frame_tok, arc->next_tok_id, frame,
                  arc_ilabels[j], arc_olabels[j], arc_weights[j], arc->acoustic_cost);
  fast_store32(arcs_apr_d + i, &apr_arc);
}


// Set start index in the buffer of the next frame
DEVICE void LatticeProcessor::SetNextSidx(int* sidx_buf, int32 size,
    int32 frame) {
  assert(frame >= 0);
  int32 cur_sidx = sidx_buf[(frame)];
  sidx_buf[(frame + 1)] = cur_sidx + size;
}

// Get the active token indexed by a uint64 pair (frame, idx), stored in void* p
// the details of the pair can be referred to LatLink::LatLink()
DEVICE Token* LatticeProcessor::GetActiveToken(void* p, bool check,
    int32 iframe) const {
  int32 frame, id;
  DECODE_TOK_IDX_PAIR(frame, id, (uint64)p);
  if (check) assert(frame == iframe || frame == iframe - 1);
  return GetActiveToken(frame, id, check);
}

// Get the active token indexed by a uint64 pair (frame, idx)
// the details of the pair can be referred to LatLink::LatLink()
DEVICE Token* LatticeProcessor::GetActiveToken(int32 frame, int32 id_pack,
    bool check) const {

  int32 cur_sidx = toks_bpr_fr_sidx_d[frame];
  int32 id = id_pack & ((1 << 31) - 1);
  assert(cur_sidx + id < toks_buf_before_pr_size);
  Token* tok = toks_bpr_d + cur_sidx + id;
  /*
  if (check) {
    assert(tok->frame == frame);
  }
  */
  return tok;
}

// Get the active token indexed by a uint64 pair (frame, idx)
// the details of the pair can be referred to LatLink::LatLink()
DEVICE Token* LatticeProcessor::GetActiveTokenByExactId(int32 frame,
    int32 id_exact, bool check) const {
  Token* tok = toks_bpr_d + id_exact;

  if (check) {
    if (id_exact < toks_bpr_fr_sidx_d[frame]) CUDA_PRINTF("h %i %i\n", id_exact,
          toks_bpr_fr_sidx_d[frame]);
    if (id_exact >= toks_bpr_fr_sidx_d[frame + 1]) CUDA_PRINTF("t %i %i\n", id_exact,
          toks_bpr_fr_sidx_d[frame + 1]);
    assert(toks_bpr_fr_sidx_d[frame] <= id_exact &&
           id_exact < toks_bpr_fr_sidx_d[frame + 1]);
  }

  return tok;
}

// Get the active arc indexed by a uint64 pair (frame, idx)
// the vector memory and the start index of each frame are kept in LatticeProcessor
DEVICE LatLinkCompact* LatticeProcessor::GetActiveArc(int32 frame,
    int32 id) const {
  int32 cur_sidx = arcs_bpr_fr_sidx_d[(frame)];
  assert(cur_sidx + id < arcs_buf_before_pr_size);
  LatLinkCompact* arc = arcs_bpr_d + cur_sidx + id;
  return arc;
}

// Size of items in the frame, it is obtained from an accumulate number array
DEVICE int32 LatticeProcessor::GetSize(int* acc_len, int32 frame) const {
  int32 size = acc_len[(frame) + 1] - acc_len[(frame)];
  assert(size >= 0 && size <= arcs_buf_before_pr_size);
  return size;
}

// used in PruneLatticeForFrame()
DEVICE void LatticeProcessor::UpdateModifiedFlags(
  volatile int32 **modified0, volatile int32 **modified1,
  volatile int32 **modified2, int cnt, int32 *modified_d) {
  *modified0 = modified_d + cnt % 3;
  *modified1 = modified_d + (cnt + 1) % 3;
  *modified2 = modified_d + (cnt + 2) % 3;
}

// The parallel lattice pruning is based on the algorithm in
// LatticeFasterDecoder::PruneActiveTokens
// with necessary modifications for GPU parallelization:
// i) parallelize the iterative updating of nodes and arcs over GPU
// threads; ii) use a global arc vector to replace the linked lists in
// the old implementation, for its lack of random access features to
// enable parallel access; iii) implement the extra cost updating as
// an atomic operation to eliminate write conflicts among threads.
// When a lattice arc is pruned, we do not physically remove
// the arc, as memory allocation is expensive. Instead, we do a
// final merging step to aggregate all remaining arcs using thread
// parallelism
// We do not prune lattice nodes because: i) we need a static mapping
// for each arc to trace the previous and the next nodes before
// and after D2H memory copy. We use frame index t and vector
// index i to trace a node, thus node positions in the vector cannot
// be changed. ii) the lattice is constructed in CPU by iterating
// remaining arcs, thus nodes are implicitly pruned. iii) node D2H
// copy is done in each frame asynchronously, which does not introduce overheads.
DEVICE void LatticeProcessor::PruneLatticeForFrame(int32 frame,
    bool merge, BaseFloat lattice_beam, int32 verbose) {
  int32 prev_cidx;
  int32 c = 0;
  int32 rank0 = threadIdx.x == 0 && blockIdx.x == 0 ? 1 : 0;
  volatile int32 *modified0;
  volatile int32 *modified1;
  volatile int32 *modified2;
  int32 cnt = 0;
  UpdateModifiedFlags(&modified0, &modified1, &modified2, cnt, modified_d);
  if (rank0 && verbose > 3) CUDA_PRINTF("%i %i\n", c++, GetSize(toks_bpr_fr_sidx_d,
                                          frame - 1)); // size before pruning
  {
    // initialize
    int32 tid = threadIdx.x + blockIdx.x * blockDim.x;
    int32 size = GetSize(toks_bpr_fr_sidx_d, frame - 1);
    for (; tid < size; tid += gridDim.x * blockDim.x) {
      Token* tok = GetActiveToken(frame - 1, tid, true);
      tok->extra_cost = FLT_MAX;
    }
    if (rank0) {
      *modified0 = 1;
      *modified1 = 0;
      *modified2 = 0;
      prev_cidx = *arcs_apr_used_d;
    }
    // wait for i) last iteration(frame+1) finish ii) finish initialization
    grid_sync(barrier_);
  }

  // iteratively updates extra costs of nodes and arcs until they stop changing,
  while (cnt++ < 10 && *modified0 != 0) {
    // triple buffer to eliminate a grid sync after *modified1 = 0;
    UpdateModifiedFlags(&modified0, &modified1, &modified2, cnt, modified_d);
    // till now, threads are using modified0 & modified2, so we clear
    // *modified1 here as it won't be used before grid sync in the very below
    if (rank0) *modified1 = 0;
    // wait for every thread to enter while, which slow down by 2% here
    //grid_sync(barrier_);

    int32 tid = threadIdx.x + blockIdx.x * blockDim.x;
    int32 size = GetSize(arcs_bpr_fr_sidx_d, frame);
    for (; tid < size; tid += gridDim.x * blockDim.x) {
      LatLinkCompact* link = GetActiveArc(frame, tid);
      int32 frame_tok = link->IsEmitArc() ? frame - 1 : frame;
      Token* next_tok = GetActiveToken(frame, link->next_tok_id, true);
      Token* tok = GetActiveToken(frame_tok, link->GetPrevTokId(), true);
      // extra cost is defined as the difference between the best
      // cost including the current arc and the best overall path.
      BaseFloat link_extra_cost = next_tok->extra_cost +
                                  ((tok->cost_ + link->acoustic_cost + arc_weights[link->arc_id])
                                   - next_tok->cost_);
      if (!isnan(link_extra_cost) && link_extra_cost <= lattice_beam) {
        // not prune out
        if (link_extra_cost < -1) {// debug
          CUDA_PRINTF("%i %f %f %f %f %f\n", frame, next_tok->extra_cost, tok->cost_,
                      link->acoustic_cost, arc_weights[link->arc_id], next_tok->cost_);
          link_extra_cost = lattice_beam / 2;
        }
        if (link_extra_cost < tok->extra_cost) {
          atomic_min(&tok->extra_cost, link_extra_cost);
          if (*modified0 == 0) atomicAdd((int32 *)modified0, 1);
        }
      }
    }
    grid_sync(barrier_);
    if (rank0 && verbose > 3) CUDA_PRINTF("%i %i\n", c++, cnt);
  }

  // final aggregate remaining arcs
  {
    int32 tid = threadIdx.x + blockIdx.x * blockDim.x;
    int32 size = GetSize(arcs_bpr_fr_sidx_d, frame);
    for (; tid < size; tid += gridDim.x * blockDim.x) {
      LatLinkCompact* link = GetActiveArc(frame, tid);
      int32 frame_tok = link->IsEmitArc() ? frame - 1 : frame;
      Token* next_tok = GetActiveToken(frame, link->next_tok_id, true);
      Token* tok = GetActiveToken(frame_tok, link->GetPrevTokId(), true);
      BaseFloat link_extra_cost = next_tok->extra_cost +
                                  ((tok->cost_ + link->acoustic_cost + arc_weights[link->arc_id])
                                   - next_tok->cost_);
      if (!isnan(link_extra_cost) && link_extra_cost <= lattice_beam) {
        // not pruned out
        if (merge) {
          AddArc(link, frame);
          // link->acoustic_cost=HIP_NAN_F;
          // don't need to delete it in original lattice
        }
      }
    }
    grid_sync(barrier_);
  }

  /*
  { // we do not prune lattice node
    // update tok
    int32 tid=threadIdx.x+blockIdx.x*blockDim.x;
    int32 size=GetSize(toks_bpr_fr_sidx_d,frame);
    for (;tid<size;tid+=gridDim.x*blockDim.x) {
      Token* tok=GetActiveToken(frame-1,tid);
      if (tok->extra_cost==FLT_MAX)
        tok->tot_cost=HIP_NAN_F; // prune
    }
  }
  */

  // get size
  if (merge && rank0) {
    int& size_arc_of_frame = arcs_apr_fr_size_d[frame];
    size_arc_of_frame = *arcs_apr_used_d - prev_cidx;
    if (verbose > 3 || (size_arc_of_frame == 0
                        && frame != 0)) CUDA_PRINTF("PR %i %i %i\n", frame,
                              GetSize(arcs_bpr_fr_sidx_d, frame), size_arc_of_frame);
  }
  // grid_sync(barrier_);
}

// copy accumulated arcs after lattice pruning till the given frame
// after obtaining the copy size, copy the buffer asynchronously
void LatticeProcessor::CopyArcsToHost(int32 frame, hipStream_t st) {
  int32 sz;
  hipMemcpy(arcs_apr_used_h, arcs_apr_used_d,
             sizeof(int32), hipMemcpyDeviceToHost);
  // TODO: optimize out above overhead
  // one possibility is we can copy static length
  // by assuming ESTIMATED_PRUNE_RATIO parts are remained
  // sz=sizeof(LatLink)*(arcs_buf_before_pr_size*ESTIMATED_PRUNE_RATIO);

  sz = sizeof(LatLink) * (*arcs_apr_used_h); // use exact count
  hipMemcpyAsync(arcs_apr_h, arcs_apr_d,
                  sz, hipMemcpyDeviceToHost, st);
  sz = sizeof(int32) * (frame + 1) * (1);
  hipMemcpyAsync(arcs_apr_fr_size_h, arcs_apr_fr_size_d,
                  sz, hipMemcpyDeviceToHost, st);
  // clear arcs_apr_used_d in GPU during next call of pruning
}

// copy accumulated toks till the given frame
// after obtaining the copy size, copy the buffer asynchronously
void LatticeProcessor::CopyToksToHost(int32 frame, hipStream_t st) {
  int32 sz;
  // include frame 0 count and the total count in the last element
  assert(frame <= prune_interval); // the max size of toks_bpr_fr_sidx_h
  sz = sizeof(int32) * (frame + 1 + 1) * (1);
  hipMemcpy(toks_bpr_fr_sidx_h, toks_bpr_fr_sidx_d,
             sz, hipMemcpyDeviceToHost);
  sz = sizeof(Token) * (toks_bpr_fr_sidx_h[frame + 1]);
  assert(sz); // assume we have obtain the total count
  hipMemcpyAsync(toks_bpr_h, toks_bpr_d,
                  sz, hipMemcpyDeviceToHost, st);
}

// get back the host data address which can be used in CPU lattice processing
void LatticeProcessor::GetHostData(Token** toks_buf, int** toks_fr_sidx,
                                LatLink** arcs_buf, int** arcs_fr_size) {
  *toks_fr_sidx = toks_bpr_fr_sidx_h;
  *toks_buf = toks_bpr_h;
  *arcs_fr_size = arcs_apr_fr_size_h; // prune_interval len
  *arcs_buf = arcs_apr_h; // start of prune_interval len arcs
}

  CudaLatticeFasterDecoder::CudaLatticeFasterDecoder(const CudaFst &fst, const CudaLatticeFasterDecoderConfig &config): fst_(fst), beam_(config.beam),
  bytes_cudaMalloc(0), max_tokens(config.max_tokens) {
    int max_token = config.max_tokens; // for CUB

    // Comments about variables are in the .h file

    hipStreamCreate(&compute_st);
    hipStreamCreate(&copy_st);

    hipEventCreate(&loglikelihood_evt);
    hipEventCreate(&loglikelihood_processed_evt);
    //hipEventCreate(&q_token_from_narcs_evt);

    hipMalloc(&d_curr_token, sizeof(int));
    hipMalloc(&d_q_token_from, sizeof(int));
    hipMalloc(&d_q_token_to, sizeof(int));
    hipMalloc(&d_q_token_end, sizeof(int));
    hipMalloc(&d_q_lat_end, sizeof(int));

    hipMalloc(&d_q_token_from_narcs, sizeof(int));
    hipHostMalloc(&h_q_token_from_narcs, sizeof(int));
  
    hipMalloc(&d_allToken, config.max_tokens * sizeof(StateId));
    hipMalloc(&d_allTokenInfo, config.max_tokens * sizeof(InfoToken));

    hipHostMalloc(&h_q_token_from_size, sizeof(int));  

    // TODO move back to params
    int max_token_frame = 5000000;
    // we could use same pointer
    hipMalloc(&d_degrees_scan, max_token_frame * sizeof(int));
    hipMalloc(&d_block_sums_scan, (max_token_frame / COMPUTE_DEGREES_DIMX + 2)* sizeof(int)); 
    hipMalloc(&d_q_arc_offset, max_token_frame * sizeof(int));

    hipMalloc(&loglikelihoods_d, sizeof(BaseFloat)*(fst_.max_ilabel+1));  
    hipMalloc(&next_loglikelihoods_d, sizeof(BaseFloat)*(fst_.max_ilabel+1));  
    hipHostMalloc(&loglikelihoods_h, sizeof(BaseFloat)*(fst_.max_ilabel+1));  


    hipMalloc(&d_state_cost,sizeof(uint64)*fst_.numStates);

    hipHostMalloc(&h_reached_final, sizeof(int));

    // TODO use directly pinned, no device mem
    // TODO hardcoded params
    hipMalloc(&d_reversed_path, 50000 * sizeof(int)); // TODO pinned
    h_reversed_path = (int*)malloc(50000 * sizeof(int));

    hipMalloc(&d_cutoff, sizeof(float));
    
    hipMalloc(&d_path_size, sizeof(int));
    hipMalloc(&d_n_CTA_done, sizeof(int));

    hipMalloc((void**)&d_dbg_tok_num,1*sizeof(int32)); 
    hipMalloc((void**)&d_barrier,1*sizeof(int32)); 
    hipMemset(d_dbg_tok_num, 0, sizeof(int));
    hipMemset(d_barrier, 0, sizeof(int));

    // for lattice
    int bytes_cuda_malloc += lattice_processor_.Allocate(config.max_tokens_per_frame,
                       config.max_lat_arc_per_frame, config.prune_interval,
                       config.max_tokens, config.max_arcs, fst_);
    lat_arcs_buf_ = lattice_processor_.GetDeviceArcsBpr();
    for (int32 i = 0; i < LAT_BUF_SIZE; i++)
      hipStreamCreateWithFlags(&stream_lat[i], hipStreamNonBlocking);

    cudaCheckError();
  }

  CudaLatticeFasterDecoder::~CudaLatticeFasterDecoder() {
        printf("CUDA DECODER DESTRUCTOR TODO\n");
      // TODO
  }

  void CudaLatticeFasterDecoder::InitDecoding() {
    printf("CUDA DECODER InitDecoding\n");


    InitLookup();

    StateId start_state = fst_.Start();
    KALDI_ASSERT(start_state != fst::kNoStateId);

    cudaCheckError();
    InfoToken it_init;
    it_init.cost = StdWeight::One().Value();
    it_init.prev_token = INT_MIN;
    it_init.arc_idx = -1;

    hipMemcpy(d_allToken, &start_state, sizeof(StateId), hipMemcpyHostToDevice);
    hipMemcpy(d_allTokenInfo, &it_init, sizeof(InfoToken), hipMemcpyHostToDevice);

    uint64 packv = pack(it_init.cost, 0);
    // We simulate a regular execution for the first iteration
    hipMemcpy(&d_state_cost[start_state], &packv, sizeof(uint64), hipMemcpyHostToDevice);

    hipMemset(d_curr_token, 0, sizeof(int));
    hipMemset(d_q_token_from, 0, sizeof(int));

    // Init state is in queue
    int one = 1;
    hipMemcpy(d_q_token_to, &one, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_q_token_end, &one, sizeof(int), hipMemcpyHostToDevice);
    *h_q_token_from_size = 1;

    float cutoff = FLT_MAX;
    hipMemcpy(d_cutoff, &cutoff, sizeof(float), hipMemcpyHostToDevice);

    hipMemset(d_n_CTA_done, 0, sizeof(int));

    hipMemset(d_dbg_tok_num, 0, sizeof(int));
    hipMemset(d_barrier, 0, sizeof(int));
    
    cudaCheckError();

    debug_max_narcs = 0;
    num_frames_decoded_ = -1; // do num_frames_decoded_++ in ComputeLogLikelihoods

    ProcessNonemitting();
 }


// Used before first frame
__global__ void init_lookup_kernel(uint64 *d_state_cost, int size) {
    for(int idx = blockIdx.x*blockDim.x + threadIdx.x;
            idx < size;
            idx += blockDim.x*gridDim.x) {
        d_state_cost[idx]  = pack(FLT_MAX,-1);
    }
}

void CudaLatticeFasterDecoder::InitLookup() {
    int nstates = fst_.numStates;


    dim3 grid,block;
    block.x = 256;
    grid.x = DIV_ROUND_UP(nstates, block.x);

    init_lookup_kernel<<<grid,block>>>(d_state_cost, nstates);
}

typedef CudaLatticeFasterDecoder::StateId StateId;

// Used to reset lookup table between frames
// Using the queue to reset only the values needed
// Also takes care of resetting cutof
DEVICE void reset_lookup_kernel(StateId *d_q, int *d_q_offset, int *d_q_end, uint64 *d_state_cost, float *d_cutoff, int *d_dbg_tok_num, int frame, int* d_q_token_from_narcs, bool reset=true) {
    int q_offset = *d_q_offset;
    int q_end = *d_q_end; 

    // Avoiding a kernel call just to reset the cutoff
    if(blockIdx.x == 0 && threadIdx.x == 0) {
        CUDA_PRINTF(2,"5 %d %d %d %d %f %d\n", frame-reset, q_end- q_offset,*d_dbg_tok_num, !reset, *d_cutoff, *d_q_token_from_narcs); 
        //reset shows the last iter is emit or not
        *d_dbg_tok_num = 0;
    }
    if (reset) {
    for(int idx = q_offset + blockIdx.x*blockDim.x + threadIdx.x;
            idx < q_end;
            idx += blockDim.x*gridDim.x) {

        StateId state = d_q[idx];

        d_state_cost[state]  = pack(FLT_MAX, -1);
    }
        if(blockIdx.x == 0 && threadIdx.x == 0) *d_cutoff = FLT_MAX;
    }


}

void CudaLatticeFasterDecoder::ResetLookup(bool reset) {
    int size = *h_q_token_from_size;

    dim3 grid,block;
    block.x = 256;
    grid.x = DIV_ROUND_UP(size, block.x);

    assert(0);
    //reset_lookup_kernel<<<grid,block,0,compute_st>>>(d_allToken, d_q_token_from, d_q_token_to, d_state_cost, d_cutoff, d_dbg_tok_num, num_frames_decoded_, d_q_token_from_narcs, reset);
}

bool CudaLatticeFasterDecoder::Decode(DecodableInterface *decodable) {
    assert(0);
    return true;
}
void CudaLatticeFasterDecoder::AdvanceDecoding(DecodableInterface *decodable,
        int32 max_num_frames) {
    KALDI_ASSERT(num_frames_decoded_ >= 0 &&
        "You must call InitDecoding() before AdvanceDecoding()");
    assert(0);

    nvtxRangePop();
}


  void CudaLatticeFasterDecoder::ComputeLogLikelihoods(DecodableInterface *decodable) {
    nvtxRangePushA("ComputeLogLikelihoods");

    //computes log likelihoods for the next frame - check order
    hipEventSynchronize(loglikelihood_processed_evt);
    hipEventSynchronize(loglikelihood_evt);
    std::swap(next_loglikelihoods_d, loglikelihoods_d);
    num_frames_decoded_++; 
    int32 frame = num_frames_decoded_;

    decodable->ComputeLogLikelihoods(loglikelihoods_h,frame,fst_.max_ilabel+1);

    //copying in another stream to overlap transfer with compute
    hipMemcpyAsync(next_loglikelihoods_d, loglikelihoods_h, sizeof(BaseFloat)*(fst_.max_ilabel+1), hipMemcpyHostToDevice,
    copy_st);

    hipEventRecord(loglikelihood_evt, copy_st);

    nvtxRangePop();
  }



void CudaLatticeFasterDecoder::InitParams(ExpandArcParams &params, uint* d_arc_offsets, bool is_emitting) {
    params.d_q = d_allToken; 
    params.d_q_info = d_allTokenInfo;

    params.d_q_token_from = d_q_token_from;
    params.d_q_token_to = d_q_token_to;
    params.d_q_token_end = d_q_token_end;

    params.d_degrees_scan = d_degrees_scan; 

    params.d_q_arc_offset = d_q_arc_offset;
    params.arc_ilabels = fst_.arc_ilabels_d;
    params.d_q_token_from_narcs = d_q_token_from_narcs;
    params.h_q_token_from_narcs = h_q_token_from_narcs;
 
    params.arc_weights = fst_.arc_weights_d; 
    params.arc_nextstates = fst_.arc_nextstates_d; 
    params.d_cutoff = d_cutoff;
    params.beam = beam_;
    params.d_loglikelihoods= loglikelihoods_d;
    params.d_lookup = d_state_cost;
    params.is_emitting = is_emitting;

    params.d_curr_token = d_curr_token;
    params.h_q_token_from_size = h_q_token_from_size;
    params.d_n_CTA_done = d_n_CTA_done;
    params.d_dbg_tok_num = d_dbg_tok_num;
    params.barrier=d_barrier;
    params.frame = num_frames_decoded_;
    params.d_arc_offsets = d_arc_offsets;
    params.d_block_sums_scan = d_block_sums_scan;
    params.d_q_lat_end = d_q_lat_end;
    params.lattice_processor = lattice_processor_;
}

bool CudaLatticeFasterDecoder::ProcessToken(unsigned int *d_arc_offsets,
                        bool is_emitting) {

    ExpandArcParams params;
    InitParams(params, d_arc_offsets, is_emitting);


    // Compute degrees, reduce by key, apply cutoff
    // Compute first part of the prefix sums of the degrees
    // At the end of that step, the kernel
    // set the value of h_q_token_from_narcs
    // (the number of arcs in the current queue processed)
    // TODO rename to something more explicit
    ComputeDegrees(params);
   
    if (params.is_emitting) {
        // finalize lattice processing of the last frame
        LatticeProcessingPerFrame(num_frames_decoded_-1);
    }
    
    // Recording an event to signal h_q_token_from_narcs 
    // as ready to use 
    //hipEventRecord(q_token_from_narcs_evt, compute_st);

    // last time we use the lookup for old_q is in compute degrees
    //ResetLookup(is_emitting); 
    /*
    if(is_emitting) {
        InitLookup();
    }
    */

    // Finalize the scan 
    // partial scans + block offsets -> global scan
    // If we want to speed up the binary search in expand
    // This is where we can compute lower and upper bound 
    // on the fly
    //FinalizeDegreesScan();
    
    // We need d_q_token_from_narcs to be ready
    //hipEventSynchronize(q_token_from_narcs_evt);
            // TODO
    //hipMemcpy(&h_old_q_narcs , d_q_token_from_narcs, sizeof(int), hipMemcpyDeviceToHost); //TODO


    bool done = false;
    if(!params.is_emitting) {
        NonEmittingLongTail(d_arc_offsets, params); 

        cudaCheckError();

        // Persistent kernel finishes the job
        done = true;
    }
    else {
        ExpandArcs(1e5, params); // TODO
    }
    if (params.is_emitting) hipEventRecord(loglikelihood_processed_evt, compute_st);

    cudaCheckError();
    return done;
}


void CudaLatticeFasterDecoder::ProcessEmitting() {
    nvtxRangePushA("ProcessEmitting");
    
    // Using emitting arc offsets
    ProcessToken(fst_.e_offsets_d, true); 

    cudaCheckError();
    nvtxRangePop();
}

  void CudaLatticeFasterDecoder::ProcessNonemitting() {
    nvtxRangePushA("ProcessNonemitting");

    // While not done, call it
    while(!ProcessToken(fst_.ne_offsets_d, false));

    cudaCheckError();
    nvtxRangePop();
  }


// TODO use struct for params, 
// large # of args slow things down

/*

This kernel is responsible for :

1) Read a token from the input queue [from, to[
2) Compute the outgoing degree of that token.next_state. For that :
   -> If that token is suboptimal (cutoff, best_cost), degree = 0
   -> Otherwise, we set degree using CSR graph

The distinction between emitting / non emitting depends on the argument passed
as "d_q_arc_offset"

3) Compute prefix sums of those degrees within the block :
    -> We store those "local prefix sums" in d_degrees_scan. Another kernel will finish the job
    -> We save the sum of all degrees in that block (block_sums)

4) The last block alive compute the prefix sums of block_sums. 
    -> We save it, it will be needed to compute global_scan
    -> We now have the total number of arcs overall, we save it to h_q_token_from_narcs

*/


DEVICE void compute_degrees_kernel(StateId *d_q, InfoToken *d_q_info, const int *d_q_token_from, const int
  *d_q_token_to, int *d_degrees_scan, unsigned int
  *d_offsets, uint64 *d_state_cost, BaseFloat *d_cutoff, int *d_q_arc_offset,
  int *d_block_sums, int *d_block_sums_scan,  int * h_q_token_from_narcs, int *d_q_token_from_narcs, int *d_n_CTA_done, int *d_dbg_tok_num) {

       typedef hipcub::BlockScan<int, COMPUTE_DEGREES_DIMX> BlockScan;
        __shared__ typename BlockScan::TempStorage temp_storage;
        __shared__ typename BlockScan::TempStorage temp_storage_scan;

        __shared__ int blk_scan_offset;
        __shared__ int is_last_CTA;

        __shared__ int new_q_block_off; // for lat

        int queue_offset = *d_q_token_from;
        int queue_end = *d_q_token_to;
        int queue_size = queue_end - queue_offset;

        BaseFloat cutoff = *d_cutoff;
        //if ( threadIdx.x==0 && blockIdx.x==0) CUDA_PRINTF("1 %d %d %d %f\n", queue_size, queue_offset, queue_end, *d_cutoff);

        for(int block_offset = blockDim.x*blockIdx.x;
                block_offset < queue_size;
                block_offset += gridDim.x*blockDim.x) {
            int idx = queue_offset + block_offset + threadIdx.x;
            int degree = 0;
            int has_successor=0, new_q_idx_block;


            InfoToken &tok = params.d_q_info[idx];
            StateId state_idx;
            BaseFloat cost;

            if(idx < queue_end) {
                state_idx = tok.GetStateId(params.arc_nextstates_d);
                cost = tok.cost_;
                if(cost < cutoff) {
                    int ptr= unpack_ptr(d_state_cost[state_idx]);
                    if(ptr == idx) {
                        int start = d_offsets[state_idx];
                        int end = d_offsets[state_idx+1];
                        degree = end - start;
                        d_q_arc_offset[idx-queue_offset] = start;
                        if (d_dbg_tok_num) atomicAdd(d_dbg_tok_num, 1);
                        has_successor++;
                    }
                }
            }
            // for lattice
            BlockScan(temp_storage_scan).ExclusiveSum(has_successor, new_q_idx_block); // we could merge the reduce and
            //the scan

            if(threadIdx.x == (COMPUTE_DEGREES_DIMX - 1)) {
                int total_block = new_q_idx_block + has_successor; // exclusive sum
                new_q_block_off = atomicAdd(params.d_q_lat_end, total_block); // TODO
            }

            // for compute degreee
            int scan;
            BlockScan(temp_storage).ExclusiveSum(degree, scan);

            // hide this sync for lat after the blockscan above
            //__syncthreads(); // newQueue_block_off + we'll reuse temp_storage_scan + global cutoff
            int new_q_index = new_q_block_off + new_q_idx_block;
            if(has_successor) {
                // store lat
                params.lat_arcs_buf_[new_q_index].Copy(LatLinkCompact(tok.prev_token_, 
                    params.is_emitting? params.frame-1:params.frame,
                       idx, params.frame, tok.GetAcousticAndInitExtraCost(), tok.arc_idx_));                
            }

            if(idx < queue_end)
                d_degrees_scan[idx-queue_offset] = scan;

            if(threadIdx.x == (COMPUTE_DEGREES_DIMX-1)) {
                d_block_sums[block_offset/COMPUTE_DEGREES_DIMX] = (scan + degree); // scan is exclusive 
            }

            // end of this block
            if((block_offset + gridDim.x*blockDim.x) < queue_end) {
                // if there's another iteration, we'll reuse temp_storage
                __syncthreads();
            }
        }

        if(threadIdx.x == 0) {
            int old = atomicAdd(d_n_CTA_done, 1);
            blk_scan_offset = 0; // will be used if last CTA, avoiding a second sync
            is_last_CTA = (old == (gridDim.x -1));
        }

        __syncthreads(); // is_last_CTA + temp_storage reuse if last CTA

        if(is_last_CTA) {
                // The last block alive takes care of scan of block sums 
                __threadfence();
                if(threadIdx.x == 0) {
                    *d_n_CTA_done = 0;
                }

                // following value can be different than gridDim.x
                int total_blk_val = (queue_size + COMPUTE_DEGREES_DIMX -1) / COMPUTE_DEGREES_DIMX;

                for(int blk_idx_off = 0;
                    blk_idx_off < total_blk_val;
                    blk_idx_off += blockDim.x) {
                    int blk_idx = blk_idx_off + threadIdx.x;

                    int blk_sum = (blk_idx < total_blk_val) ? d_block_sums[blk_idx] : 0;

                    int blk_scan;
                    BlockScan(temp_storage).ExclusiveSum(blk_sum, blk_scan);
                    blk_scan += blk_scan_offset; 
                
                    if(blk_idx < total_blk_val) {
                        d_block_sums_scan[blk_idx] = blk_scan;
                    }
                    
                    __syncthreads(); // blk_scan_offset + reuse temp_storage
                    if(threadIdx.x == (COMPUTE_DEGREES_DIMX-1)) {
                        int total = blk_scan + blk_sum;
                        blk_scan_offset = total;
                    }

                }

            if(threadIdx.x == 0) {
                *d_q_token_from_narcs = blk_scan_offset; // TODO
                *h_q_token_from_narcs = blk_scan_offset; // TODO
            }
        }
  }

/*

Part 2 of the scan. Computes global prefix sum with block prefix sum and block offsets

If we want to speed up expand, we can compute lower and upper bound to restrain 
the binary search in expand
This can be done on the fly here, and removes main bottleneck of expand
Not done for now, because expand is fast enough

*/
DEVICE void finalize_degrees_scan_kernel(int *d_scan, int *d_blk_scan, const int *d_q_token_from, const int
  *d_q_token_to) {

        int q_off = *d_q_token_from;
        int q_end = *d_q_token_to;
        int q_size = q_end - q_off;

        for(int idx = blockDim.x*blockIdx.x + threadIdx.x;
                idx < q_size;
                idx += blockDim.x*gridDim.x) {

            int blk_idx = idx / blockDim.x;
            int blk_scan_offset = d_blk_scan[blk_idx]; // we rely on L1 for this one, avoiding syncs

            d_scan[idx] += blk_scan_offset;
        }

 }

typedef CudaLatticeFasterDecoder::ExpandArcParams ExpandArcParams; 

// for lattice
DEVICE void lattice_process_per_frame(ExpandArcParams &params) {
  // TODO call from __global__
  // process lattice before allocate new toks to TokenState
  params.lattice_processor.CollectToksPerFrame(d_q_token_end, params.frame-1);
  // accumulatively store lattice arcs
  params.lattice_processor.CollectArcsPerFrame(d_q_lat_end, params.frame-1);
}

void __global__ compute_degrees_with_reset_kernel(ExpandArcParams params, bool reset=true) {
  compute_degrees_kernel(params.d_q, params.d_q_info,params.d_q_token_from, 
      params.d_q_token_to, params.d_degrees_scan, params.d_arc_offsets, 
      params.d_lookup, params.d_cutoff, params.d_q_arc_offset, 
      params.d_block_sums_scan, params.d_block_sums_scan,  params.h_q_token_from_narcs, params.d_q_token_from_narcs, 
      params.d_n_CTA_done, params.d_dbg_tok_num);
  grid_sync(params.barrier);
  reset_lookup_kernel(params.d_q, params.d_q_token_from, params.d_q_token_to, params.d_lookup, params.d_cutoff, params.d_dbg_tok_num, params.frame, params.d_q_token_from_narcs, reset);
  finalize_degrees_scan_kernel(params.d_degrees_scan, params.d_block_sums_scan, params.d_q_token_from, params.d_q_token_to);
  if (params.is_emitting) lattice_process_per_frame(params);
}
  void CudaLatticeFasterDecoder::FinalizeDegreesScan() {
      dim3 grid,block;
      block.x = COMPUTE_DEGREES_DIMX;
      grid.x = DIV_ROUND_UP(*h_q_token_from_size, block.x);

      assert(0);
      //finalize_degrees_scan_kernel<<<grid,block,0,compute_st>>>(d_degrees_scan, d_block_sums_scan, d_q_token_from, d_q_token_to); 
  }
 
  void CudaLatticeFasterDecoder::ComputeDegrees(const ExpandArcParams &params) {
    dim3 grid,block;
    block.x = COMPUTE_DEGREES_DIMX;
    grid.x = DIV_ROUND_UP(*h_q_token_from_size, block.x);

    compute_degrees_with_reset_kernel<<<grid,block,0,compute_st>>>(params, params.is_emitting);
    cudaCheckError();
  }

   

__forceinline__ __device__ int binsearch_maxle(const int *vec, const int val, int low, int high) {
    while(true) {
        if(low == high)
            return low; //we know it exists
        if((low + 1) == high)
            return (vec[high] <= val) ? high : low;

        int mid = low + (high- low) / 2;

        if(vec[mid] > val)
            high = mid-1;
        else
            low = mid;
    }
}


// Temporary used for cutoff - will be removed
__device__ float fatomicMin(float *addr, float val)

{
  BaseFloat minval = *addr;
  while (val < minval) {  // if my value is less than minimum
    minval = val;         // update the minimum to my value locally
    // write minimum and read back value
    val = atomicExch(addr, val);
  } // if the new value is < the minimum I wrote I need to try again.
  return minval;
}


/*

This kernel propagates arcs from the current queue [from,to[
to the new queue [to,end[

The main bottleneck is the first binary search. 
If we want to remove that bottleneck, cf comments on FinalizeScan


TODO merge reduce and scan for code simplicity + remove syncs

The last block alive moves the queues indexes :
new from is old to
new to is new end
new end stays new end


*/

void __global__ get_cutoff(ExpandArcParams params, BaseFloat set = 0) {
    typedef hipcub::BlockScan<int, EXPAND_ARCS_DIMX> BlockScan;
    typedef hipcub::BlockReduce<BaseFloat, EXPAND_ARCS_DIMX> BlockReduce;
    
    __shared__ typename BlockScan::TempStorage temp_storage_scan;
    __shared__ typename BlockReduce::TempStorage temp_storage_reduce;

    __shared__ int new_q_block_off;
    __shared__ BaseFloat global_cutoff;
 
    const int total_narcs = *params.d_q_token_from_narcs;
    const int old_q_offset = *params.d_q_token_from;
    const int old_q_size = *params.d_q_token_to - old_q_offset;

    if (set) {
      if ( threadIdx.x==0 && blockIdx.x==0) *params.d_cutoff = set;
      return;
    }
    if(threadIdx.x == 0) {
        global_cutoff = *params.d_cutoff;
    }

    // Keeping the whole CTA alive, we'll have syncs
    for(int block_offset = blockDim.x*blockIdx.x;
            block_offset < total_narcs;
            block_offset += gridDim.x*blockDim.x) {

        int th_idx = block_offset + threadIdx.x;
        bool valid_input = (th_idx < total_narcs);

        StateId prev_state;
        BaseFloat total_cost = FLT_MAX;
        int arc_idx;
        StateId arc_next_state;
        int q_idx;

        if(valid_input) {
            //we can do better than that
            q_idx = old_q_offset + binsearch_maxle(params.d_degrees_scan, th_idx, 0, old_q_size-1); 
            
            int lower_bound = params.d_degrees_scan[q_idx - old_q_offset];
            prev_state = params.d_q[q_idx];

            int arc_offset_start = params.d_q_arc_offset[q_idx - old_q_offset];
            arc_idx = arc_offset_start + (block_offset + threadIdx.x - lower_bound);

            arc_next_state = params.arc_nextstates[arc_idx];
            BaseFloat arc_weight = params.arc_weights[arc_idx];
            
            int arc_ilabel = params.is_emitting ? params.arc_ilabels[arc_idx] : 0;

            BaseFloat acoustic_cost = (arc_ilabel != 0) ? -params.d_loglikelihoods[arc_ilabel] : 0.0; 

            BaseFloat old_tok_cost = params.d_q_info[q_idx].cost;

            total_cost = acoustic_cost + arc_weight + old_tok_cost;

            BaseFloat next_state_cost = unpack_cost(params.d_lookup[arc_next_state]);
            if(total_cost > next_state_cost) {
                total_cost = FLT_MAX;
                valid_input = false; 
            } 
        }
        
        BaseFloat thread_cutoff = (total_cost < FLT_MAX) ? (total_cost + params.beam) : FLT_MAX;
        BaseFloat new_block_cutoff = BlockReduce(temp_storage_reduce).Reduce(thread_cutoff, hipcub::Min());

        if(threadIdx.x == 0) {
            if(new_block_cutoff < global_cutoff) {
                BaseFloat new_global_cutoff = fatomicMin(params.d_cutoff, new_block_cutoff);
                new_global_cutoff = min(new_global_cutoff, new_block_cutoff);
                global_cutoff = new_global_cutoff;
            }
        }
        
        __syncthreads(); //BlockReduce

    }
}
void __global__ expand_arcs_kernel(ExpandArcParams params) {
    typedef hipcub::BlockScan<int, EXPAND_ARCS_DIMX> BlockScan;
    typedef hipcub::BlockReduce<BaseFloat, EXPAND_ARCS_DIMX> BlockReduce;
    
    __shared__ typename BlockScan::TempStorage temp_storage_scan;
    __shared__ typename BlockReduce::TempStorage temp_storage_reduce;

    __shared__ int new_q_block_off;
 
    const int total_narcs = *params.d_q_token_from_narcs;
    const int old_q_offset = *params.d_q_token_from;
    const int old_q_size = *params.d_q_token_to - old_q_offset;

    //if ( threadIdx.x==0 && blockIdx.x==0) CUDA_PRINTF("5.0 %d %d %f %f\n", old_q_size, total_narcs, *params.d_cutoff, params.d_loglikelihoods[0]);
 
    // Keeping the whole CTA alive, we'll have syncs
    for(int block_offset = blockDim.x*blockIdx.x;
            block_offset < total_narcs;
            block_offset += gridDim.x*blockDim.x) {

        int th_idx = block_offset + threadIdx.x;
        bool valid_input = (th_idx < total_narcs);

        StateId prev_state;
        BaseFloat total_cost = FLT_MAX;
        int arc_idx;
        StateId arc_next_state;
        int q_idx;

        BaseFloat acoustic_cost = 0;
        if(valid_input) {
            //we can do better than that
            q_idx = old_q_offset + binsearch_maxle(params.d_degrees_scan, th_idx, 0, old_q_size-1); 
            
            int lower_bound = params.d_degrees_scan[q_idx - old_q_offset];
            prev_state = params.d_q[q_idx];

            int arc_offset_start = params.d_q_arc_offset[q_idx - old_q_offset];
            arc_idx = arc_offset_start + (block_offset + threadIdx.x - lower_bound);

            arc_next_state = params.arc_nextstates[arc_idx];
            BaseFloat arc_weight = params.arc_weights[arc_idx];
            
            int arc_ilabel = params.is_emitting ? params.arc_ilabels[arc_idx] : 0;

            acoustic_cost = (arc_ilabel != 0) ? -params.d_loglikelihoods[arc_ilabel] : 0.0; 
            BaseFloat next_state_cost = unpack_cost(params.d_lookup[arc_next_state]);

            BaseFloat old_tok_cost = params.d_q_info[q_idx].cost;

            total_cost = acoustic_cost + arc_weight + old_tok_cost;

            if(total_cost > next_state_cost) {
                total_cost = FLT_MAX;
                valid_input = false; 
            } 
        }
       
        BaseFloat cutoff = *params.d_cutoff;

        int has_successor = (total_cost < cutoff && valid_input) ? 1 : 0;

        int new_q_idx_block;

        BlockScan(temp_storage_scan).ExclusiveSum(has_successor, new_q_idx_block); // we could merge the reduce and
        //the scan

        if(threadIdx.x == (EXPAND_ARCS_DIMX - 1)) {
            int total_block = new_q_idx_block + has_successor; // exclusive sum
            new_q_block_off = atomicAdd(params.d_q_token_end, total_block);
        }

        __syncthreads(); // newQueue_block_off + we'll reuse temp_storage_scan + global cutoff

        int new_q_index = new_q_block_off + new_q_idx_block;

        if(has_successor) {
            //params.d_q[new_q_index] = arc_next_state;
            params.d_q_info[new_q_index].Copy(InfoToken(total_cost, acoustic_cost, q_idx, arc_idx));
        }
        if(has_successor) {
            // reduce, not atomic (no return)
            atomicMin((unsigned long long *)&params.d_lookup[arc_next_state], (unsigned long long)pack(total_cost, new_q_index));
        }
    }


    // Last block alive moves queue 

    if(threadIdx.x == 0) {
        int old = atomicAdd(params.d_n_CTA_done, 1);
        if(old == (gridDim.x -1)) {
            // The last block alive takes care of preparing for next iter
            __threadfence(); // we want last value of d_q_token_end
            int final_end = *params.d_q_token_end;

            *params.h_q_token_from_size = final_end - *params.d_q_token_to;

            *params.d_n_CTA_done = 0;
            *params.d_q_token_from = *params.d_q_token_to;
            *params.d_q_token_to = final_end;

            if(params.is_emitting) {
                // Saving position of curr_token for this frame
                // We'll need to reset d_q_token_from for next frame
                *params.d_curr_token = *params.d_q_token_from;
            }
        }
    }

}

void CudaLatticeFasterDecoder::ExpandArcs(int nthreads, const ExpandArcParams &params) {
    dim3 grid,block;
    block.x = EXPAND_ARCS_DIMX;
    grid.x = DIV_ROUND_UP(nthreads, block.x);

    get_cutoff<<<grid,block,0,compute_st>>>(params);
    expand_arcs_kernel<<<grid,block,0,compute_st>>>(params);
}



// Reached final kernel
__global__ void reached_final_kernel(StateId *d_q, const int *d_q_token_from, const int *d_q_token_to, BaseFloat *final, float fst_zero, int *h_reached_final) {
    int q_offset = *d_q_token_from;
    int q_end = *d_q_token_to;

    for(int idx = q_offset + blockDim.x*blockIdx.x + threadIdx.x;
            idx < q_end;
            idx += blockDim.x*gridDim.x) {

       StateId state = d_q[idx];
       float final_val = final[state]; 

       if(final_val != fst_zero) {
            *h_reached_final = 1; // we could exit
       }
    }

}

  bool CudaLatticeFasterDecoder::ReachedFinal() const {
      dim3 grid, block;
      block.x = 256;
      grid.x = DIV_ROUND_UP(*h_q_token_from_size, block.x);

      reached_final_kernel<<<grid,block>>>(d_allToken, d_q_token_from, d_q_token_to, fst_.final_d, StdWeight::Zero().Value(), h_reached_final);
      hipDeviceSynchronize(); //TODO...

      return *h_reached_final;
  }



// Used to find best costs.
// TODO Needs to be rewritten

#define FILL_COSTS_DIMX 256
__global__ void fill_costs_kernel(StateId *d_q, InfoToken *d_q_it, const int *d_q_token_from, const int *d_q_token_to,
uint64 *d_state_cost, BaseFloat *d_final, bool final) {
    int q_offset = *d_q_token_from;
    int q_end = *d_q_token_to;

    for(int idx = q_offset + blockIdx.x*blockDim.x + threadIdx.x;
            idx < q_end;
            idx += blockDim.x*gridDim.x) {
        BaseFloat cost = d_q_it[idx].cost;
        
        if(final) {
            StateId state = d_q[idx];
            cost += d_final[state];
        }
        
        d_state_cost[idx-q_offset] = pack(cost,idx);
    }

}


void CudaLatticeFasterDecoder::GetBestCost(BaseFloat *min, int *arg, bool isfinal) const {
    dim3 grid, block;
    block.x = FILL_COSTS_DIMX;

    grid.x = DIV_ROUND_UP(*h_q_token_from_size, block.x);

    // TODO using lookup as float buffer for now - NEED TO CHANGE
    fill_costs_kernel<<<grid,block>>>(d_allToken, d_allTokenInfo,
    d_q_token_from, d_q_token_to, d_state_cost, fst_.final_d, isfinal);

    hipcub::KeyValuePair<int, uint64> *d_argmin;
    hipMalloc(&d_argmin, sizeof(hipcub::KeyValuePair<int, int>));
    
    void *d_temp_storage_amin = NULL;
    size_t temp_storage_amin_bytes = 0;

    int max_t = max_tokens;
    hipcub::DeviceReduce::ArgMin(d_temp_storage_amin, temp_storage_amin_bytes, d_state_cost, d_argmin, *h_q_token_from_size);
    hipMalloc(&d_temp_storage_amin, temp_storage_amin_bytes);

    hipcub::DeviceReduce::ArgMin(d_temp_storage_amin, temp_storage_amin_bytes, d_state_cost, d_argmin, *h_q_token_from_size);

    hipcub::KeyValuePair<int, uint64> h_argmin;

    hipMemcpy(&h_argmin, d_argmin, sizeof(hipcub::KeyValuePair<int, int>), hipMemcpyDeviceToHost);
   

    hipFree(d_temp_storage_amin);
    hipFree(d_argmin);

    //InitLookup(); // reset lookup

    //*min = orderedUIntToFloat(h_argmin.value);
    *min = -10; // TODO switch back to real value once new kernel ready
    *arg = h_argmin.key;
}

  BaseFloat CudaLatticeFasterDecoder::FinalRelativeCost() const {
    if(*h_q_token_from_size == 0)
        return FLT_MAX;

      BaseFloat best_cost;
      int arg_best;
      GetBestCost(&best_cost, &arg_best, false);


      BaseFloat best_cost_final;
      int arg_best_final;
      GetBestCost(&best_cost_final, &arg_best_final, true);

      return (best_cost_final - best_cost);
  }

// brutal - one thread, multiple global memory load. But avoids a massive memcpy D2H
// Will disappear with better memory management 
void __global__ get_best_path_kernel(int best_token_idx_in_all_tokens, StateId *d_all_tokens, InfoToken
*d_all_tokens_info, int *d_reversed_path, int *path_size) {

    int tok_idx = best_token_idx_in_all_tokens;
    int idx = 0;

    while(tok_idx != INT_MIN) {
        int state = d_all_tokens[tok_idx];
        int arc_idx = d_all_tokens_info[tok_idx].arc_idx;
        d_reversed_path[idx++] = arc_idx;

        int old_tok_idx = tok_idx; 
        tok_idx = d_all_tokens_info[tok_idx].prev_token;
        assert(old_tok_idx > tok_idx);
            
    }
    
    *path_size = idx;
}

// Outputs an FST corresponding to the single best path
  // through the lattice.
  bool CudaLatticeFasterDecoder::GetBestPath(Lattice *fst_out, bool use_final_probs) const {
      nvtxRangePushA("GetBestPath");

      BaseFloat best_cost;
      int arg_best;
      GetBestCost(&best_cost, &arg_best, false);

      BaseFloat best_cost_final;
      int arg_best_final;
      GetBestCost(&best_cost_final, &arg_best_final, true);

      bool isfinal = ReachedFinal();

      int h_curr_token_offset;
      hipMemcpy(&h_curr_token_offset, d_q_token_from, sizeof(int), hipMemcpyDeviceToHost);

      int h_best_token_idx = isfinal ? arg_best_final : arg_best; 
      h_best_token_idx += h_curr_token_offset;

      /*
    printf("is final = %i \n", isfinal);
    printf("curr token off=%i \n", h_curr_token_offset);
    printf("best token idx=%i \n", h_best_token_idx);
    printf("final costs : %f  final = %f \n", best_cost, best_cost_final);
    printf("final costs idx : %i  final idx = %i \n", arg_best, arg_best_final);
    */

    hipMemset(d_path_size, 0, sizeof(int));

    get_best_path_kernel<<<1,1>>>(h_best_token_idx, d_allToken, d_allTokenInfo, d_reversed_path, d_path_size);

    hipDeviceSynchronize();

    
    int h_path_size;
    hipMemcpy(&h_path_size, d_path_size, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_reversed_path, d_reversed_path, h_path_size * sizeof(int), hipMemcpyDeviceToHost);
    

    fst_out->DeleteStates();
     
     // We can assert first state equals to root
    
    StateId cur_state = fst_out->AddState();
    fst_out->SetStart(cur_state);

    // -1 for 0-indexing, -1 for ignoring starting arc
    for (int i = h_path_size-1-1; i >= 1; i--) {
      int arc_idx = h_reversed_path[i];
      LatticeArc arc(fst_.arc_ilabels_h[arc_idx], fst_.arc_olabels_h[arc_idx], LatticeWeight(fst_.arc_weights_h[arc_idx], 0), fst_.arc_nextstates_h[arc_idx]);

      arc.nextstate = fst_out->AddState();
      fst_out->AddArc(cur_state, arc);
      cur_state = arc.nextstate;
    }

    if (isfinal && use_final_probs)
      fst_out->SetFinal(cur_state,
          LatticeWeight(fst_.Final(fst_.arc_nextstates_h[h_reversed_path[0]]), 0.0));
    else
      fst_out->SetFinal(cur_state, LatticeWeight::One());

    fst::RemoveEpsLocal(fst_out);

    nvtxRangePop();
      return true;
  }


// Wrote for single CTA

/*

Persistent kernel

Used to avoid calling multiple "heavy lifting" kernels for the tail of non emitting
(lots of iterations with small number of arcs)

Code is greatly simplified because we can have only one CTA alive

Repeat until new queue empty:
    1) Computes degrees (cf ComputeDegrees) 
    2) Compute scan
    3) Expand arcs

1 and 2 are not done on the first iteration, because it's already done
(by corresponding kernels)

At the end, this kernel finalize the computation for current frame,
setting the queue [from,to[ to the complete curr_token queue
so that it's ready for next ProcessEmitting

We could optimize and speed up this kernel
It will only gives us a better latency for 1 stream, which is low enough
Instead, we let it compute while we use the GPU for other streams
This kernel only uses one block, and is a free rider on the GPU

*/


__launch_bounds__(NONEM_LT_DIMX, 1)
__global__ void process_nonem_longtail(unsigned int *d_arc_offsets, 
                                ExpandArcParams params, int* d_dbg_tok_num) {

    typedef hipcub::BlockScan<int, NONEM_LT_DIMX> BlockScan;
    typedef hipcub::BlockReduce<float, NONEM_LT_DIMX> BlockReduce;

    __shared__ typename BlockScan::TempStorage temp_storage_scan;
    __shared__ typename BlockReduce::TempStorage temp_storage_reduce;

    __shared__ int total_narcs;
    __shared__ int new_q_end;
    __shared__ int new_q_lat_end;

    BaseFloat cutoff;
    int old_q_offset = *params.d_q_token_from;
    int new_q_offset = *params.d_q_token_to;

    if(threadIdx.x == 0) {
        new_q_end = *params.d_q_token_end;
        new_q_lat_end = *params.d_q_lat_end;
        total_narcs = *params.d_q_token_from_narcs;
    }

    __syncthreads();

    int old_q_size = new_q_offset - old_q_offset;  // move to end
    
    cutoff = *params.d_cutoff;
    //if ( threadIdx.x==0 && blockIdx.x==0) CUDA_PRINTF("3 %f %d\n",cutoff, old_q_size);
    
    // We'll switch queue at the beg of the loop
    // Cleaner that way - we need the offsets ready for
    // the global updates at the very end of this kernel
    new_q_offset = old_q_offset;
   
    bool first = true;
    int total_at=0;

    while(old_q_size > 0) {
        // Step 0 : move queues        
        old_q_offset = new_q_offset;
        new_q_offset = new_q_end;

        if(!first) {

            if(threadIdx.x == 0)  {
                total_narcs = 0;
            }

            // Step 1 : compute_degrees
            for(int local_q_idx = threadIdx.x;
                    local_q_idx < old_q_size;
                    local_q_idx += blockDim.x) {

                int global_q_idx = old_q_offset + local_q_idx;

                InfoToken &tok = params.d_q_info[global_q_idx];
                StateId state = tok.GetStateId(params.arc_nextstates_d);
                BaseFloat cost = tok.cost;

                int degree = 0;
                int has_successor = 0, new_q_idx_block;
                if(cost < cutoff) {
                    int ptr = unpack_ptr(params.d_lookup[state]);

                    if(ptr == global_q_idx) {
                        int start = d_arc_offsets[state];
                        int end = d_arc_offsets[state+1];
                        degree = end - start;
                        params.d_q_arc_offset[local_q_idx] = start;
                        if (d_dbg_tok_num) atomicAdd(d_dbg_tok_num, 1);
                        has_successor++;
                    }
                }

                params.d_degrees_scan[local_q_idx] = degree;
                
                // for lattice
                BlockScan(temp_storage_scan).ExclusiveSum(has_successor, new_q_idx_block); // we could merge the reduce and

                if (has_successor) {
                    int new_q_index = new_q_lat_end + new_q_idx_block;
                    //the scan                
                    params.lat_arcs_buf_[new_q_index].Copy(LatLinkCompact(tok.prev_token_, params.frame,
                           global_q_idx, params.frame, // should be non_emitting
                           tok.GetAcousticAndInitExtraCost(), tok.arc_idx_));
                }
                if(threadIdx.x == (NONEM_LT_DIMX - 1)) {
                    int total_in_block = new_q_idx_block + has_successor; // exclusive sum
                    new_q_lat_end += total_in_block;
                }                
                __syncthreads(); // temp_storage_scan
            }

            /*
            __syncthreads();
            if ( threadIdx.x==0 && blockIdx.x==0) {
            for (int i=0; i<old_q_size ;i++) {
                  printf("%d ",params.d_degrees_scan[i]);
                }
                printf(" : %d\n",total_narcs);
            }
            */

            // Step 2 : Scan

            for(int block_off = 0;
                    block_off < old_q_size;
                    block_off += blockDim.x) {

                int local_q_idx = block_off + threadIdx.x;

                int degree = (local_q_idx < old_q_size) 
                    ? params.d_degrees_scan[local_q_idx]
                    : 0;
                int lscan;
                BlockScan(temp_storage_scan).ExclusiveSum(degree, lscan);
                int scan = lscan + total_narcs;

                if(local_q_idx < old_q_size)
                    params.d_degrees_scan[local_q_idx] = scan;

                if (local_q_idx==0) assert(lscan==0);
                __syncthreads(); // total_narcs
                if(threadIdx.x == (NONEM_LT_DIMX-1)) {
                    int total_in_block = lscan + degree;
                    total_narcs += total_in_block;
                }
            }

        } else {
            first = false;    
        }

        if ( threadIdx.x==0 && blockIdx.x==0) {
          CUDA_PRINTF(4,"4.0 %f %d %d\n",cutoff, old_q_size, *d_dbg_tok_num);
          total_at+=*d_dbg_tok_num;
          *d_dbg_tok_num=0;
            /*
               for (int i=0; i<old_q_size ;i++) {
                  printf("%d ",params.d_degrees_scan[i]);
                }
                printf(" : %d\n",total_narcs);
                */
                
        }

        __syncthreads(); //total_narcs

        // Step 3 : expand arcs

        for(int block_offset = 0;
                block_offset < total_narcs;
                block_offset += blockDim.x) {

            int th_idx = block_offset + threadIdx.x;
            bool valid_input = (th_idx < total_narcs);

            BaseFloat total_cost = FLT_MAX;
            int arc_idx;
            StateId arc_next_state;
            int q_idx, local_q_idx=-1;

            if(valid_input) {
                //we can do better than that
                local_q_idx = binsearch_maxle(params.d_degrees_scan, th_idx, 0, old_q_size-1); // get from token idx

                int lower_bound = params.d_degrees_scan[local_q_idx];
                int arc_offset_start = params.d_q_arc_offset[local_q_idx];
                q_idx = old_q_offset + local_q_idx;

                arc_idx = arc_offset_start + (th_idx - lower_bound);

                arc_next_state = params.arc_nextstates[arc_idx];
                BaseFloat arc_weight = params.arc_weights[arc_idx];
                BaseFloat next_state_cost = unpack_cost(params.d_lookup[arc_next_state]);
                BaseFloat old_tok_cost = params.d_q_info[q_idx].cost;

                total_cost = arc_weight + old_tok_cost;

                if(total_cost > next_state_cost) {
                    total_cost = FLT_MAX;
                    valid_input = false; 
                } 
            }

            BaseFloat thread_cutoff = (total_cost < FLT_MAX) ? (total_cost + params.beam) : FLT_MAX;
            BaseFloat new_block_cutoff = BlockReduce(temp_storage_reduce).Reduce(thread_cutoff, hipcub::Min());

            /*
            if(threadIdx.x == 0) {
                if(new_block_cutoff < cutoff) {
                    cutoff = new_block_cutoff;
                }
            }
            */

            int has_successor = (total_cost < cutoff && valid_input) ? 1 : 0;

            int new_q_idx_block, new_q_index;

            BlockScan(temp_storage_scan).ExclusiveSum(has_successor, new_q_idx_block);

            if(has_successor) {
                new_q_index = new_q_end + new_q_idx_block;
                
                //params.d_q[new_q_index] = arc_next_state;
                params.d_q_info[new_q_index].Copy(InfoToken(total_cost, 0, q_idx, arc_idx));
            }
            if(has_successor) 
                atomicMin((unsigned long long *)&params.d_lookup[arc_next_state], (unsigned long long )pack(total_cost, new_q_index));
 
            //if (has_successor || arc_idx == 8299288 || arc_idx == 8508243|| local_q_idx ==6 || local_q_idx ==7)
            //    printf("%d:%d:%f ", arc_idx, local_q_idx, total_cost);

            if(threadIdx.x == (NONEM_LT_DIMX - 1)) {
                int total_in_block = new_q_idx_block + has_successor; // exclusive sum
                new_q_end += total_in_block;

            }
        }

        __syncthreads(); // new_q_end

        old_q_size = new_q_end - new_q_offset; 

    }

    if(threadIdx.x == 0) {
        // Next step is ProcessEmitting of next frame, from is currToken_offset
        *params.d_q_token_from = *params.d_curr_token; 
        *params.d_q_token_to = new_q_end;
        *params.d_q_token_end = new_q_end;
        *params.d_q_lat_end = new_q_lat_end; 
        // TODO *params.d_q_lat_end = ?

        //*params.d_cutoff = cutoff;
    if ( threadIdx.x==0 && blockIdx.x==0) CUDA_PRINTF(3,"4 %f %d %d\n",cutoff, *params.d_q_token_to-*params.d_q_token_from, total_at);

        *params.h_q_token_from_size = new_q_end - *params.d_q_token_from;
    }

}
  
void CudaLatticeFasterDecoder::NonEmittingLongTail(unsigned int *d_arc_offsets, 
                                const ExpandArcParams &params) {

    dim3 grid,block;
    block.x = NONEM_LT_DIMX;
    grid.x = 1; // it is designed for the long tail
    process_nonem_longtail<<<grid,block,0,compute_st>>>(d_arc_offsets, params, d_dbg_tok_num);
}


// for lattice
// GPU lattice prune and copy the processed lattice nodes and arcs to host
void CudaLatticeDecoder::FinalProcessLattice(Token** toks_buf, int** toks_fr_sidx,
    LatLink** arcs_buf, int** arcs_fr_size) {
  PUSH_RANGE("FinalProcessLattice", 3)

  // TODO: last frame lattice processing
  {
    ExpandArcParams params;
    num_frames_decoded_++;
    InitParams(params, fst_.e_offsets_d, true);
    num_frames_decoded_--; // TODO
    ComputeDegrees(params); // lattice proc inner this func
  }
  
  hipStreamSynchronize(compute_st); // after fini comp. we can start copy
  // copy unpruned toks to host
  lattice_processor_.CopyToksToHost(num_frames_decoded_, stream_lat[0]);
  // GPU lattice pruning
  PruneActiveTokens(compute_st, compute_st, config_.lat_fraction);
  // copy the TokenState vector in the last frame, used by ComputeFinalCosts()
  CU_SAFE_CALL(hipGetLastError());
  
  hipStreamSynchronize(compute_st); // wait for lattice pruning
  // copy pruned lattice arcs to host
  lattice_processor_.CopyArcsToHost(num_frames_decoded_, stream_lat[1]);
  // wait for all streams finishing
  hipStreamSynchronize(stream_lat[0]);
  hipStreamSynchronize(stream_lat[1]);
  // get host data from lattice_processor_, used by CPU lattice processing
  lattice_processor_.GetHostData(toks_buf, toks_fr_sidx,
                              arcs_buf, arcs_fr_size);
  CU_SAFE_CALL(hipGetLastError());

  KALDI_VLOG(1) << "Average tokens number, total frame: "
                << (*toks_fr_sidx)[num_frames_decoded_ + 1] / num_frames_decoded_
                << ", " << num_frames_decoded_;
  POP_RANGE
}

void CudaLatticeDecoder::PruneActiveTokens(hipStream_t wait_st,
    hipStream_t run_st, BaseFloat gpu_ratio) {
  // we launch 64 threads as a block, i.e. 2 cooperative_groups
  // in cuda kernel of dynamic load balancing. more details are described there
  // we use a static launch size to reduce the kernel launch time 30us->10us
  dim3 threads(64, 1);
  dim3 blocks(DIV_ROUND_UP(total_threads * gpu_ratio, (threads.x * threads.y)));
  hipStreamSynchronize(wait_st);
  if (config_.verbose > 1) KALDI_LOG << "PruneActiveTokens, # of blocks: " <<
                                       blocks.x << std::endl;
  processTokens_params params;
  InitParams(&params);
  _prune_active_tokens <<< blocks, threads, 0, run_st>>>(params);
}
} // end namespace kaldi.

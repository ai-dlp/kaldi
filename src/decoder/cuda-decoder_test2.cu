#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "base/timer.h"

#ifdef __HIPCC__
  #define HOST __host__
  #define DEVICE __device__

#else
  #define HOST
  #define DEVICE
#endif

#define cudaCheckError() {                                          \
        hipError_t e=hipGetLastError();                                 \
        if(e!=hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    }


DEVICE void acquire_semaphore(volatile int *lock){
  while (atomicCAS((int *)lock, 0, 1) != 0);
  }

DEVICE void release_semaphore(volatile int *lock){
  //*lock = 0;
  atomicExch((unsigned int*)lock,0u);
  __threadfence();
  }

  template<int blockDimx, int blockDimy>
  inline DEVICE void myadd(int *ret, volatile int *mutex) {
    (*(ret+threadIdx.x))++;
    (*(mutex+threadIdx.x*blockIdx.x))++;
  }
  template<int blockDimx, int blockDimy>
  inline DEVICE void myread(int *ret, volatile int *mutex) {
    (*(ret+threadIdx.x));
    (*(mutex+threadIdx.x*blockIdx.x));
  }


  template<int blockDimx, int blockDimy>
  inline DEVICE void myadd2(int *ret, volatile int *mutex) {
    if (threadIdx.x==0) {
    acquire_semaphore((int*)(mutex+threadIdx.x*blockIdx.x));
    (*(ret+threadIdx.x))++;
    release_semaphore((int*)(mutex+threadIdx.x*blockIdx.x));
    }
  }
  template<int blockDimx, int blockDimy>
  inline DEVICE void myadd0(int *ret, volatile int *mutex) {
    acquire_semaphore((int*)(mutex));
    (*(ret))++;
    release_semaphore((int*)(mutex));
  }

  __global__ void callmyread(int *ret, int *mutex) {
  //myadd2<32,2>(ret, mutex);
  myread<320,1>(ret, mutex);
  //myadd<32,2>(ret, mutex);
  }

  __global__ void callmyadd(int *ret, int *mutex) {
  //myadd2<32,2>(ret, mutex);
  myadd<100,1>(ret, mutex);
  //myadd<32,2>(ret, mutex);
  }
int main() {
  //int blocks=200;
  int blocks=3;
  //int blocks=7;
  int *mutex=0;
  int *ret=0, ret_h=0;
  int n =1e2;
  int32_t device;
  kaldi::Timer timer;

  hipGetDevice(&device);
  /*
  hipMallocManaged((void**)&v_man,sizeof(int)*n);  
  hipMallocManaged((void**)&ret,sizeof(int)*n);  
  hipMemset(v_man, 0,sizeof(int)*n);
  hipMemset(ret, 0,sizeof(int)*n);
  hipMemAdvise(v_man,sizeof(int)*n,hipMemAdviseSetPreferredLocation,device);
  hipMemAdvise(ret,sizeof(int)*n,hipMemAdviseSetPreferredLocation,device);
  hipMemPrefetchAsync(v_man,sizeof(int)*n,device);  //force pages to allocate now
  hipMallocManaged((void**)&vv,sizeof(int)*n);  
  hipMemAdvise(vv,sizeof(int)*n,hipMemAdviseSetPreferredLocation,device);
  hipMemPrefetchAsync(vv,sizeof(int)*n,device);  //force pages to allocate now
*/

  int test=100;
  int *v_man, *vv, *v_man_f[100], *v_man_h;
  for (int i=0; i<test;i++) {
    hipMalloc((void**)&v_man_f[i],sizeof(int)*n);  
  }
  std::cout<<sizeof(int)*n*test<<std::endl;
  hipHostMalloc((void**)&v_man_h,sizeof(int)*n*test);  
  hipMalloc((void**)&v_man,sizeof(int)*n*test);  
  cudaCheckError();
  hipStream_t stream_comp;
  hipStreamCreateWithFlags(&stream_comp, hipStreamNonBlocking);

  double t1,t2_1,t2_2;
  timer.Reset();
  for (int i=0; i<test;i++)
    hipMemcpyAsync(v_man_h+i*n,v_man_f[i],sizeof(int)*n,hipMemcpyDeviceToHost, stream_comp);
  cudaCheckError();
  hipStreamSynchronize(stream_comp);
  t1=timer.Elapsed();

  timer.Reset();
  for (int i=0; i<test;i++)
    hipMemcpyAsync(v_man+i*n,v_man_f[i],sizeof(int)*n,hipMemcpyDeviceToDevice, stream_comp);
  hipStreamSynchronize(stream_comp);
  t2_1=timer.Elapsed();
  hipMemcpyAsync(v_man_h,v_man,sizeof(int)*n*test,hipMemcpyDeviceToHost, stream_comp);
  cudaCheckError();
  hipStreamSynchronize(stream_comp);
  t2_2=timer.Elapsed();

  std::cout<<t1<<" "<<t2_1<< " "<<t2_2<<std::endl;
  
  return 0;
#if 0
  callmyadd<<<100,320>>>(ret, v_man);
  cudaCheckError();

  //time
  int s0=0;
  timer.Reset();
  for (int i=0;i<n;i++)  s0+=v_man[i];
  t0=timer.Elapsed();
  timer.Reset();
  for (int i=0;i<n;i++)  v_man[i];
  t0_1=timer.Elapsed();

  /*
  //time
  timer.Reset();
  for (int i=0;i<n;i++) int k=v_man[i];
  //time
  timer.Reset();
  for (int i=0;i<n;i++) int k=v_man[i];
 */
  hipMemPrefetchAsync(v_man, sizeof(int)* n,hipCpuDeviceId,NULL);  

   //time
  timer.Reset();
  for (int i=0;i<n;i++) int k=v_man[i];
  t1=timer.Elapsed();
 
  callmyadd<<<100,320>>>(ret, v_man);
  cudaCheckError();

  //time
  timer.Reset();
  for (int i=0;i<n;i++) int k=v_man[i];
  t2=timer.Elapsed();

  callmyadd<<<100,320>>>(ret, v_man);
  cudaCheckError();

  //time
  timer.Reset();
  for (int i=0;i<n;i++) int k=v_man[i];
  t2_1=timer.Elapsed();

  callmyread<<<300,320>>>(ret, v_man);
  cudaCheckError();

  //time
  timer.Reset();
  for (int i=0;i<n;i++) int k=v_man[i];
  t2_2=timer.Elapsed();


  callmyadd<<<100,320>>>(ret, v_man);
  cudaCheckError();

  hipMemPrefetchAsync(v_man, sizeof(int)* n,hipCpuDeviceId,NULL); 

  //time
  int s=0;
  timer.Reset();
  for (int i=0;i<n;i++)  s+=v_man[i];
  t3=timer.Elapsed();

  hipMemPrefetchAsync(vv, sizeof(int)* n,hipCpuDeviceId,NULL); 

  //time
  timer.Reset();
  for (int i=0;i<n;i++)  v_man[i];
  t4=timer.Elapsed();

  hipMemPrefetchAsync(v_man, sizeof(int)* n,hipCpuDeviceId,NULL); 

  //time
  timer.Reset();
  for (int i=0;i<n;i++)  v_man[i];
  t5=timer.Elapsed();

  std::cout << " nop "<<t0<< " re "<<t0_1<<" p "<<t1<<" mod "<<t2 <<" re "<<t2_1 <<" read "<<t2_2<<" pf "<<t3<<" other "<<t4<<" repf "<<t5<< " "<<ret[0] <<" "<<v_man[0]<<" "<<s0<<" "<<s<<std::endl;
#endif

  //hipFree(v_man);
  //hipFree(vv);
}

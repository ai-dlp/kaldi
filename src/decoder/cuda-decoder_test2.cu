#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "base/timer.h"
#include <hip/hip_runtime.h>
#include "cuda_device_runtime_api.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#ifdef __HIPCC__
  #define HOST __host__
  #define DEVICE __device__

#else
  #define HOST
  #define DEVICE
#endif

#define cudaCheckError() {                                          \
        hipError_t e=hipGetLastError();                                 \
        if(e!=hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    }


DEVICE void acquire_semaphore(volatile int *lock){
  while (atomicCAS((int *)lock, 0, 1) != 0);
  }

DEVICE void release_semaphore(volatile int *lock){
  //*lock = 0;
  atomicExch((unsigned int*)lock,0u);
  __threadfence();
  }

  template<int blockDimx, int blockDimy>
  inline DEVICE void myadd(int *ret, volatile int *mutex) {
    (*(ret+threadIdx.x))++;
    (*(mutex+threadIdx.x*blockIdx.x))++;
  }
  template<int blockDimx, int blockDimy>
  inline DEVICE void myread(int *ret, volatile int *mutex) {
    (*(ret+threadIdx.x));
    (*(mutex+threadIdx.x*blockIdx.x));
  }


  template<int blockDimx, int blockDimy>
  inline DEVICE void myadd2(int *ret, volatile int *mutex) {
    if (threadIdx.x==0) {
    acquire_semaphore((int*)(mutex+threadIdx.x*blockIdx.x));
    (*(ret+threadIdx.x))++;
    release_semaphore((int*)(mutex+threadIdx.x*blockIdx.x));
    }
  }
  template<int blockDimx, int blockDimy>
  inline DEVICE void myadd0(int *ret, volatile int *mutex) {
    acquire_semaphore((int*)(mutex));
    (*(ret))++;
    release_semaphore((int*)(mutex));
  }

  __global__ void callmyread(int *ret, int *mutex) {
  //myadd2<32,2>(ret, mutex);
  myread<320,1>(ret, mutex);
  //myadd<32,2>(ret, mutex);
  }

  __global__ void callmyadd(int *ret, int *mutex) {
  //myadd2<32,2>(ret, mutex);
  myadd<100,1>(ret, mutex);
  //myadd<32,2>(ret, mutex);
  }
  __global__ void memcp(int *to, int **from, int len, int size, hipStream_t st) {
    //for (int i=0;i<size;i++) 
    int i=blockIdx.x;
    hipMemcpyAsync(to+i*len/sizeof(int), from[i], len, hipMemcpyDeviceToDevice);
    //}
  }
  __global__ void memcp2(int *to, int **from, int len, int size, hipStream_t st) {
    //for (int i=0;i<size;i++) 
    int i=blockIdx.x;
    int tid = threadIdx.x;
    int sz = len/sizeof(int);
    for(; tid < sz; tid += blockDim.x) {
      to[tid+i*blockDim.x]=from[i][tid];
    }
    //hipMemcpyAsync(to+i*len/sizeof(int), from[i], len, hipMemcpyDeviceToDevice);
    //}
  }


__global__ void random(int** result, int m, int n) {
  hiprandState_t state;
  hiprand_init(0, /* the seed controls the sequence of random values that are produced */
                    0, /* the sequence number is only important with multiple cores */
                                  0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                                                &state);
  if (threadIdx.x==0){
    for (int j=0;j<m;j++)
    for (int i=0;i<n;i++)
  result[j][i] = hiprand(&state)%10;
  }
}
int main() {
  //int blocks=200;
  int blocks=3;
  //int blocks=7;
  int *mutex=0;
  int *ret=0, ret_h=0;
  int n =1e2;
  int32_t device;
  kaldi::Timer timer;

  hipGetDevice(&device);
  /*
  hipMallocManaged((void**)&v_man,sizeof(int)*n);  
  hipMallocManaged((void**)&ret,sizeof(int)*n);  
  hipMemset(v_man, 0,sizeof(int)*n);
  hipMemset(ret, 0,sizeof(int)*n);
  hipMemAdvise(v_man,sizeof(int)*n,hipMemAdviseSetPreferredLocation,device);
  hipMemAdvise(ret,sizeof(int)*n,hipMemAdviseSetPreferredLocation,device);
  hipMemPrefetchAsync(v_man,sizeof(int)*n,device);  //force pages to allocate now
  hipMallocManaged((void**)&vv,sizeof(int)*n);  
  hipMemAdvise(vv,sizeof(int)*n,hipMemAdviseSetPreferredLocation,device);
  hipMemPrefetchAsync(vv,sizeof(int)*n,device);  //force pages to allocate now
*/

  int test=100;
  int *v_man, *vv, *v_man_f[100], *v_man_h, s, **v_man_d;
  for (int i=0; i<test;i++) {
    hipMalloc((void**)&v_man_f[i],sizeof(int)*n);  
  }
  std::cout<<sizeof(int)*n*test<<std::endl;
  hipHostMalloc((void**)&v_man_h,sizeof(int)*n*test);  
  hipMalloc((void**)&v_man,sizeof(int)*n*test);  
  hipMalloc((void**)&v_man_d,sizeof(int*)*test);  
  hipMemcpy(v_man_d, v_man_f, sizeof(int*)*test, hipMemcpyHostToDevice);
  cudaCheckError();
  hipStream_t stream_comp;
  hipStream_t stream_comp2;
  hipStreamCreateWithFlags(&stream_comp, hipStreamNonBlocking);
  hipStreamCreateWithFlags(&stream_comp2, hipStreamNonBlocking);
  //for (int i=0; i<test;i++)
  random<<<1,1,0,stream_comp>>>(v_man_d, test, n);
  hipStreamSynchronize(stream_comp);
  cudaCheckError();

  double t1,t2_1,t2_2;
  hipMemset(v_man_h, 0,sizeof(int)*test*n);
  timer.Reset();
  for (int i=0; i<test;i++)
    hipMemcpyAsync(v_man_h+i*n,v_man_f[i],sizeof(int)*n,hipMemcpyDeviceToHost, stream_comp);
  cudaCheckError();
  hipStreamSynchronize(stream_comp);
  t1=timer.Elapsed();

  s=0;
  for (int i=0; i<test; i++) for (int j=0; j<n; j++) s+=v_man_h[i*n+j];
  int s1=s;


  hipMemset(v_man_h, 0,sizeof(int)*test*n);
  hipMemset(v_man, 0,sizeof(int)*test*n);

  timer.Reset();
  for (int i=0; i<test;i++)
    hipMemcpyAsync(v_man+i*n,v_man_f[i],sizeof(int)*n,hipMemcpyDeviceToDevice, stream_comp);
  hipStreamSynchronize(stream_comp);
  t2_1=timer.Elapsed();
  hipMemcpyAsync(v_man_h,v_man,sizeof(int)*n*test,hipMemcpyDeviceToHost, stream_comp);
  cudaCheckError();
  hipStreamSynchronize(stream_comp);
  t2_2=timer.Elapsed();

  s=0;
  for (int i=0; i<test; i++) for (int j=0; j<n; j++) s+=v_man_h[i*n+j];
  int s2=s;


  hipMemset(v_man_h, 0,sizeof(int)*test*n);
  hipMemset(v_man, 0,sizeof(int)*test*n);
  timer.Reset();
  //memcp2<<<100,100,0, stream_comp>>>(v_man, v_man_d, sizeof(int)*n, test, stream_comp2);
  memcp<<<100,1,0, stream_comp>>>(v_man, v_man_d, sizeof(int)*n, test, stream_comp2);
  hipStreamSynchronize(stream_comp);
  hipStreamSynchronize(stream_comp2);
  double t3_1=timer.Elapsed();
  hipMemcpyAsync(v_man_h,v_man,sizeof(int)*n*test,hipMemcpyDeviceToHost, stream_comp);
  hipStreamSynchronize(stream_comp);
  cudaCheckError();
  double t3_2=timer.Elapsed();

  s=0;
  for (int i=0; i<test; i++) for (int j=0; j<n; j++) s+=v_man_h[i*n+j];
  int s3=s;

  hipMemset(v_man_h, 0,sizeof(int)*test*n);
  hipMemset(v_man, 0,sizeof(int)*test*n);
  s=0;
  for (int i=0; i<test; i++) for (int j=0; j<n; j++) s+=v_man_h[i*n+j];
  int s1_1=s;
  timer.Reset();
  memcp2<<<100,100,0, stream_comp>>>(v_man, v_man_d, sizeof(int)*n, test, stream_comp2);
  hipStreamSynchronize(stream_comp);
  hipStreamSynchronize(stream_comp2);
  double t4_1=timer.Elapsed();
  hipMemcpyAsync(v_man_h,v_man,sizeof(int)*n*test,hipMemcpyDeviceToHost, stream_comp);
  hipStreamSynchronize(stream_comp);
  cudaCheckError();
  double t4_2=timer.Elapsed();

  s=0;
  for (int i=0; i<test; i++) for (int j=0; j<n; j++) s+=v_man_h[i*n+j];
  int s4=s;


  std::cout<<t1<<" "<<s1<<" "<<s1_1<<" "<<t2_1<< " "<<t2_2<<" "<<s2<<" "<<t3_1<<" "<<t3_2<<" "<<s3<<" "<<t4_1<<" "<<t4_2<<" "<<s4<<std::endl;
  
  return 0;
#if 0
  callmyadd<<<100,320>>>(ret, v_man);
  cudaCheckError();

  //time
  int s0=0;
  timer.Reset();
  for (int i=0;i<n;i++)  s0+=v_man[i];
  t0=timer.Elapsed();
  timer.Reset();
  for (int i=0;i<n;i++)  v_man[i];
  t0_1=timer.Elapsed();

  /*
  //time
  timer.Reset();
  for (int i=0;i<n;i++) int k=v_man[i];
  //time
  timer.Reset();
  for (int i=0;i<n;i++) int k=v_man[i];
 */
  hipMemPrefetchAsync(v_man, sizeof(int)* n,hipCpuDeviceId,NULL);  

   //time
  timer.Reset();
  for (int i=0;i<n;i++) int k=v_man[i];
  t1=timer.Elapsed();
 
  callmyadd<<<100,320>>>(ret, v_man);
  cudaCheckError();

  //time
  timer.Reset();
  for (int i=0;i<n;i++) int k=v_man[i];
  t2=timer.Elapsed();

  callmyadd<<<100,320>>>(ret, v_man);
  cudaCheckError();

  //time
  timer.Reset();
  for (int i=0;i<n;i++) int k=v_man[i];
  t2_1=timer.Elapsed();

  callmyread<<<300,320>>>(ret, v_man);
  cudaCheckError();

  //time
  timer.Reset();
  for (int i=0;i<n;i++) int k=v_man[i];
  t2_2=timer.Elapsed();


  callmyadd<<<100,320>>>(ret, v_man);
  cudaCheckError();

  hipMemPrefetchAsync(v_man, sizeof(int)* n,hipCpuDeviceId,NULL); 

  //time
  int s=0;
  timer.Reset();
  for (int i=0;i<n;i++)  s+=v_man[i];
  t3=timer.Elapsed();

  hipMemPrefetchAsync(vv, sizeof(int)* n,hipCpuDeviceId,NULL); 

  //time
  timer.Reset();
  for (int i=0;i<n;i++)  v_man[i];
  t4=timer.Elapsed();

  hipMemPrefetchAsync(v_man, sizeof(int)* n,hipCpuDeviceId,NULL); 

  //time
  timer.Reset();
  for (int i=0;i<n;i++)  v_man[i];
  t5=timer.Elapsed();

  std::cout << " nop "<<t0<< " re "<<t0_1<<" p "<<t1<<" mod "<<t2 <<" re "<<t2_1 <<" read "<<t2_2<<" pf "<<t3<<" other "<<t4<<" repf "<<t5<< " "<<ret[0] <<" "<<v_man[0]<<" "<<s0<<" "<<s<<std::endl;
#endif

  //hipFree(v_man);
  //hipFree(vv);
}

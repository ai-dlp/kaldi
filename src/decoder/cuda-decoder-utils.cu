#include "hip/hip_runtime.h"
// decoder/cuda-decoder-utils.cc

// Copyright      2018  Zhehuai Chen

// See ../../COPYING for clarification regarding multiple authors
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
// http:// www.apache.org/licenses/LICENSE-2.0
// 
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.

#include "fst/fstlib.h"
#include "cuda-decoder-utils.h"

namespace kaldi {

void get_free_memory_stat(char *prefix) {
  int32 act_gpu_id;
  hipError_t e = hipGetDevice(&act_gpu_id);
  char name[128];
  CuDevice::Instantiate().DeviceGetName(name,128,act_gpu_id);
  // get GPU memory stats
  int64 free, total; 
  std::string mem_stats;
  mem_stats = CuDevice::Instantiate().GetFreeMemory(&free, &total);
  float mem_ratio = total > 0 ? free/(float)total : 0;
  // log
  KALDI_LOG << prefix << "\tcudaSetDevice(" << act_gpu_id << "): "
            << name << "\t" << mem_ratio << " % "
            << "\t" << mem_stats;
}




// CudaFst Implementation
HOST DEVICE float CudaFst::Final(StateId state) const {
#ifdef __CUDA_ARCH__
  return final_d[state];
#else
  return final_h[state];
#endif
}
void CudaFst::Initialize(const fst::Fst<StdArc> &fst) {
  PUSH_RANGE("CudaFst constructor", 1)
  bytes_cudaMalloc = 0;
  numStates = 0;
  for ( fst::StateIterator<fst::Fst<StdArc> > iter(fst); !iter.Done();
        iter.Next()) {
    numStates++;
  }
  start = fst.Start();
  hipHostMalloc(&final_h, sizeof(float) * numStates);
  hipMalloc(&final_d, sizeof(float) * numStates);

  // allocate and initialize offset arrays
  e_offsets_h = (unsigned int *)malloc(sizeof(unsigned int) * (numStates + 1));
  ne_offsets_h = (unsigned int *)malloc(sizeof(unsigned int) * (numStates + 1));

  hipMalloc((void**)&e_offsets_d, sizeof(unsigned int) * (numStates + 1));
  bytes_cudaMalloc += sizeof(unsigned int) * (numStates + 1);
  hipMalloc((void**)&ne_offsets_d, sizeof(unsigned int) * (numStates + 1));
  bytes_cudaMalloc += sizeof(unsigned int) * (numStates + 1);

  memset(e_offsets_h, 0, sizeof(unsigned int) * (numStates + 1));
  memset(ne_offsets_h, 0, sizeof(unsigned int) * (numStates + 1));

  // iterate through states and arcs and count number of arcs per state
  e_count = 0;
  ne_count = 0;
  max_ilabel = 0;
  for (int i = 0; i < numStates; i++) {
    final_h[i] = fst.Final(i).Value();
    // count emmiting and non_emitting arcs
    for (fst::ArcIterator<fst::Fst<StdArc> > aiter(fst, i); !aiter.Done();
         aiter.Next()) {
      StdArc arc = aiter.Value();
      int32 ilabel = arc.ilabel;
      int32 olabel = arc.olabel;
      if (ilabel > max_ilabel) {
        max_ilabel = ilabel;
      }
      if (ilabel != 0) { // emitting
        e_count++;
      } else { // non-emitting
        ne_count++;
      }
    }
    ne_offsets_h[i + 1] = ne_count;
    e_offsets_h[i + 1] = e_count;
  }

  // offset ne_offsets by the number of emitting arcs
  for (int i = 0; i < numStates + 1; i++) {
    e_offsets_h[i] += 1;        // add dummy arc at the beginingg.
    ne_offsets_h[i] += e_count + 1; // add dummy arc and put e_arcs before
  }

  arc_count = e_count + ne_count + 1;
  numArcs = arc_count;

  hipMemcpyAsync(final_d, final_h, sizeof(float)*numStates, hipMemcpyHostToDevice,
                  hipStreamPerThread);
  hipMemcpyAsync(e_offsets_d, e_offsets_h, sizeof(unsigned int) * (numStates + 1),
                  hipMemcpyHostToDevice, hipStreamPerThread);
  hipMemcpyAsync(ne_offsets_d, ne_offsets_h,
                  sizeof(unsigned int) * (numStates + 1), hipMemcpyHostToDevice,
                  hipStreamPerThread);

  // Allocate non-zero arrays
  hipHostMalloc(&arc_weights_h, arc_count * sizeof(BaseFloat));
  hipHostMalloc(&arc_nextstates_h, arc_count * sizeof(StateId));
  hipHostMalloc(&arc_ilabels_h, arc_count * sizeof(int32));
  hipHostMalloc(&arc_olabels_h, arc_count * sizeof(int32));

  hipMalloc((void**)&arc_weights_d, arc_count * sizeof(BaseFloat));
  bytes_cudaMalloc += arc_count * sizeof(BaseFloat);
  hipMalloc((void**)&arc_nextstates_d, arc_count * sizeof(StateId));
  bytes_cudaMalloc += arc_count * sizeof(StateId);
  hipMalloc((void**)&arc_ilabels_d, arc_count * sizeof(int32));
  bytes_cudaMalloc += arc_count * sizeof(int32);
  hipMalloc((void**)&arc_olabels_d, arc_count * sizeof(int32));
  bytes_cudaMalloc += arc_count * sizeof(int32);

  // now populate arc data
  int e_idx = 1;        // save room for dummy arc (so start at 1)
  int ne_idx = e_count + 1; // starts where e_offsets ends

  // create dummy arc
  arc_weights_h[0] = StdWeight::One().Value();
  arc_nextstates_h[0] = fst.Start();
  arc_ilabels_h[0] = 0;
  arc_olabels_h[0] = 0;

  for (int i = 0; i < numStates; i++) {
    // count emmiting and non_emitting arcs
    for (fst::ArcIterator<fst::Fst<StdArc> > aiter(fst, i); !aiter.Done();
         aiter.Next()) {
      StdArc arc = aiter.Value();
      int idx;
      if (arc.ilabel != 0) { // emitting
        idx = e_idx++;
      } else {
        idx = ne_idx++;
      }
      arc_weights_h[idx] = arc.weight.Value();
      arc_nextstates_h[idx] = arc.nextstate;
      arc_ilabels_h[idx] = arc.ilabel;
      arc_olabels_h[idx] = arc.olabel;
    }
  }

  hipMemcpyAsync(arc_weights_d, arc_weights_h, arc_count * sizeof(BaseFloat),
                  hipMemcpyHostToDevice, hipStreamPerThread);
  hipMemcpyAsync(arc_nextstates_d, arc_nextstates_h, arc_count * sizeof(StateId),
                  hipMemcpyHostToDevice, hipStreamPerThread);
  hipMemcpyAsync(arc_ilabels_d, arc_ilabels_h, arc_count * sizeof(int32),
                  hipMemcpyHostToDevice, hipStreamPerThread);
  hipMemcpyAsync(arc_olabels_d, arc_olabels_h, arc_count * sizeof(int32),
                  hipMemcpyHostToDevice, hipStreamPerThread);

  hipStreamSynchronize(hipStreamPerThread);
  POP_RANGE
}

void CudaFst::Finalize() {
  PUSH_RANGE("CudaFst destructor", 0);
  hipHostFree(final_h);
  hipFree(final_d);
  free(e_offsets_h);
  free(ne_offsets_h);

  hipFree(e_offsets_d);
  hipFree(ne_offsets_d);

  hipHostFree(arc_weights_h);
  hipHostFree(arc_nextstates_h);
  hipHostFree(arc_ilabels_h);
  hipHostFree(arc_olabels_h);

  hipFree(arc_weights_d);
  hipFree(arc_nextstates_d);
  hipFree(arc_ilabels_d);
  hipFree(arc_olabels_d);
  POP_RANGE
}

} // end namespace kaldi.


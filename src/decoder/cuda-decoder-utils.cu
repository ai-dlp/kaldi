#include "hip/hip_runtime.h"
// decoder/cuda-decoder-utils.cc

// Copyright      2018  Zhehuai Chen

// See ../../COPYING for clarification regarding multiple authors
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//  http://www.apache.org/licenses/LICENSE-2.0
//
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.

#include "fst/fstlib.h"
#include "cuda-decoder-utils.h"

namespace kaldi {

DEVICE inline void __gpu_sync_fast(volatile int *fast_epoch) {
  __syncthreads();
  if (threadIdx.x == 0) {
    // gridDim.x-1 blocks are adding 1
    // and one block is adding 0x80000000 - (gridDim.x-1)
    // so the whole sum is 0x80000000
    int nb = 1;
    if (blockIdx.x == 0) {
      nb = 0x80000000 - (gridDim.x - 1);
    }
    int old_epoch = *fast_epoch;
    __threadfence();
    atomicAdd((int*)fast_epoch, nb);
    // wait for the sign bit to commute
    int cnt = 0;
    while (((*fast_epoch) ^ old_epoch) >= 0) ;
  }
  __syncthreads();
}
DEVICE  void __grid_sync_nv_internal(int *barrier) {
  __gpu_sync_fast((volatile int*)barrier);
}

// CudaFst Implementation
HOST DEVICE float CudaFst::Final(StateId state) const {
#ifdef __CUDA_ARCH__
  return final_d[state];
#else
  return final_h[state];
#endif
}
void CudaFst::Initialize(const fst::Fst<StdArc> &fst) {
  PUSH_RANGE("CudaFst constructor", 1)
  bytes_cudaMalloc = 0;
  numStates = 0;
  for ( fst::StateIterator<fst::Fst<StdArc> > iter(fst); !iter.Done();
        iter.Next()) {
    numStates++;
  }
  start = fst.Start();
  hipHostMalloc(&final_h, sizeof(float) * numStates);
  hipMalloc(&final_d, sizeof(float) * numStates);

  // allocate and initialize offset arrays
  e_offsets_h = (unsigned int *)malloc(sizeof(unsigned int) * (numStates + 1));
  ne_offsets_h = (unsigned int *)malloc(sizeof(unsigned int) * (numStates + 1));

  hipMalloc((void**)&e_offsets_d, sizeof(unsigned int) * (numStates + 1));
  bytes_cudaMalloc += sizeof(unsigned int) * (numStates + 1);
  hipMalloc((void**)&ne_offsets_d, sizeof(unsigned int) * (numStates + 1));
  bytes_cudaMalloc += sizeof(unsigned int) * (numStates + 1);

  memset(e_offsets_h, 0, sizeof(unsigned int) * (numStates + 1));
  memset(ne_offsets_h, 0, sizeof(unsigned int) * (numStates + 1));

  // iterate through states and arcs and count number of arcs per state
  e_count = 0;
  ne_count = 0;
  max_ilabel = 0;
  for (int i = 0; i < numStates; i++) {
    final_h[i] = fst.Final(i).Value();
    // count emmiting and non_emitting arcs
    for (fst::ArcIterator<fst::Fst<StdArc> > aiter(fst, i); !aiter.Done();
         aiter.Next()) {
      StdArc arc = aiter.Value();
      int32 ilabel = arc.ilabel;
      int32 olabel = arc.olabel;
      if (ilabel > max_ilabel) {
        max_ilabel = ilabel;
      }
      if (ilabel != 0) { // emitting
        e_count++;
      } else { // non-emitting
        ne_count++;
      }
    }
    ne_offsets_h[i + 1] = ne_count;
    e_offsets_h[i + 1] = e_count;
  }

  // offset ne_offsets by the number of emitting arcs
  for (int i = 0; i < numStates + 1; i++) {
    e_offsets_h[i] += 1;        // add dummy arc at the beginingg.
    ne_offsets_h[i] += e_count + 1; // add dummy arc and put e_arcs before
  }

  arc_count = e_count + ne_count + 1;
  numArcs = arc_count;

  hipMemcpyAsync(final_d, final_h, sizeof(float)*numStates, hipMemcpyHostToDevice,
                  hipStreamPerThread);
  hipMemcpyAsync(e_offsets_d, e_offsets_h, sizeof(unsigned int) * (numStates + 1),
                  hipMemcpyHostToDevice, hipStreamPerThread);
  hipMemcpyAsync(ne_offsets_d, ne_offsets_h,
                  sizeof(unsigned int) * (numStates + 1), hipMemcpyHostToDevice,
                  hipStreamPerThread);

  // Allocate non-zero arrays
  hipHostMalloc(&arc_weights_h, arc_count * sizeof(BaseFloat));
  hipHostMalloc(&arc_nextstates_h, arc_count * sizeof(StateId));
  hipHostMalloc(&arc_ilabels_h, arc_count * sizeof(int32));
  hipHostMalloc(&arc_olabels_h, arc_count * sizeof(int32));

  hipMalloc((void**)&arc_weights_d, arc_count * sizeof(BaseFloat));
  bytes_cudaMalloc += arc_count * sizeof(BaseFloat);
  hipMalloc((void**)&arc_nextstates_d, arc_count * sizeof(StateId));
  bytes_cudaMalloc += arc_count * sizeof(StateId);
  hipMalloc((void**)&arc_ilabels_d, arc_count * sizeof(int32));
  bytes_cudaMalloc += arc_count * sizeof(int32);
  hipMalloc((void**)&arc_olabels_d, arc_count * sizeof(int32));
  bytes_cudaMalloc += arc_count * sizeof(int32);

  // now populate arc data
  int e_idx = 1;        // save room for dummy arc (so start at 1)
  int ne_idx = e_count + 1; // starts where e_offsets ends

  // create dummy arc
  arc_weights_h[0] = StdWeight::One().Value();
  arc_nextstates_h[0] = fst.Start();
  arc_ilabels_h[0] = 0;
  arc_olabels_h[0] = 0;

  for (int i = 0; i < numStates; i++) {
    // count emmiting and non_emitting arcs
    for (fst::ArcIterator<fst::Fst<StdArc> > aiter(fst, i); !aiter.Done();
         aiter.Next()) {
      StdArc arc = aiter.Value();
      int idx;
      if (arc.ilabel != 0) { // emitting
        idx = e_idx++;
      } else {
        idx = ne_idx++;
      }
      arc_weights_h[idx] = arc.weight.Value();
      arc_nextstates_h[idx] = arc.nextstate;
      arc_ilabels_h[idx] = arc.ilabel;
      arc_olabels_h[idx] = arc.olabel;
    }
  }

  hipMemcpyAsync(arc_weights_d, arc_weights_h, arc_count * sizeof(BaseFloat),
                  hipMemcpyHostToDevice, hipStreamPerThread);
  hipMemcpyAsync(arc_nextstates_d, arc_nextstates_h, arc_count * sizeof(StateId),
                  hipMemcpyHostToDevice, hipStreamPerThread);
  hipMemcpyAsync(arc_ilabels_d, arc_ilabels_h, arc_count * sizeof(int32),
                  hipMemcpyHostToDevice, hipStreamPerThread);
  hipMemcpyAsync(arc_olabels_d, arc_olabels_h, arc_count * sizeof(int32),
                  hipMemcpyHostToDevice, hipStreamPerThread);

  hipStreamSynchronize(hipStreamPerThread);
  POP_RANGE
}

void CudaFst::Finalize() {
  PUSH_RANGE("CudaFst destructor", 0);
  hipHostFree(final_h);
  hipFree(final_d);
  free(e_offsets_h);
  free(ne_offsets_h);

  hipFree(e_offsets_d);
  hipFree(ne_offsets_d);

  hipHostFree(arc_weights_h);
  hipHostFree(arc_nextstates_h);
  hipHostFree(arc_ilabels_h);
  hipHostFree(arc_olabels_h);

  hipFree(arc_weights_d);
  hipFree(arc_nextstates_d);
  hipFree(arc_ilabels_d);
  hipFree(arc_olabels_d);
  POP_RANGE
}

} //  end namespace kaldi.

